#include "hip/hip_runtime.h"
#include "GSRBCuda.h"
#include "GSRBConstants.h"


#define DEFAULT_THRESHOLD  4000
#define BLOCKSIZE 64
#define TILESIZE 16
#define cudaCheck(x) _cudaCheck(x, #x ,__FILE__, __LINE__)

template<typename T>
void _cudaCheck(T e, const char* func, const char* call, const int line){
  if(e != hipSuccess){
    printf("\"%s\" at %d in %s\n\treturned %d\n-> %s\n", func, line, call, (int)e, hipGetErrorString(e));
    exit(EXIT_FAILURE);
  }
}

__global__ void GSRBKernel(double* phi, double* phi_new, double* rhs, double* alpha, double* beta_i,
                           double* beta_j, double* beta_k, double* lambda, int color)
{
    // TODO: Find out what i, j, k maps to in terms of blockIdx, blockDim, threadIdx
    int currentOffset = blockIdx.x + blockDim.x + threadIdx.x;

    int i, j, k;
    i = currentOffset+1;

    for (k=1; k<pencil-1; k++)
    {
        for(j=1; j<pencil-1; j++)
        {

            int ijk = i + j*pencil + k*plane;

            if ((i+j+k+color) % 2 == 0)
            {
                double helmholtz = alpha[ijk]*phi[ijk]
                                 - H2INV*(
                                       beta_i[ijk+1     ]*( phi[ijk+1     ]-phi[ijk       ] )
                                     - beta_i[ijk       ]*( phi[ijk       ]-phi[ijk-1     ] )
                                     + beta_j[ijk+pencil]*( phi[ijk+pencil]-phi[ijk       ] )
                                     - beta_j[ijk       ]*( phi[ijk       ]-phi[ijk-pencil] )
                                     + beta_k[ijk+plane ]*( phi[ijk+plane ]-phi[ijk       ] )
                                     - beta_k[ijk       ]*( phi[ijk       ]-phi[ijk-plane ] )
                                      );

                phi_new[ijk] = phi[ijk] - lambda[ijk]*(helmholtz-rhs[ijk]);
            }
        }
    }
}

void GSRBCuda(double* phi, double* phi_new, double* rhs, double* alpha, double* beta_i,
              double* beta_j, double* beta_k, double* lambda)
{
    //CUDA Buffers
    double* phi_device    ;
    double* phi_new_device;
    double* rhs_device    ;
    double* alpha_device  ;
    double* beta_i_device ;
    double* beta_j_device ;
    double* beta_k_device ;
    double* lambda_device ;

    double* tmp;

    // Init Memory on GPU
    // Cuda Memory Management
    cudaCheck(hipMalloc((void**) &phi_device    , grid * sizeof(double)));
    cudaCheck(hipMalloc((void**) &phi_new_device, grid * sizeof(double)));
    cudaCheck(hipMalloc((void**) &rhs_device    , grid * sizeof(double)));
    cudaCheck(hipMalloc((void**) &alpha_device  , grid * sizeof(double)));
    cudaCheck(hipMalloc((void**) &beta_i_device , grid * sizeof(double)));
    cudaCheck(hipMalloc((void**) &beta_j_device , grid * sizeof(double)));
    cudaCheck(hipMalloc((void**) &beta_k_device , grid * sizeof(double)));
    cudaCheck(hipMalloc((void**) &lambda_device , grid * sizeof(double)));
    cudaCheck(hipGetLastError());

    cudaCheck(hipMemcpy(phi_device    , phi    , grid * sizeof(double), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(phi_new_device, phi_new, grid * sizeof(double), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(rhs_device    , rhs    , grid * sizeof(double), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(alpha_device  , alpha  , grid * sizeof(double), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(beta_i_device , beta_i , grid * sizeof(double), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(beta_j_device , beta_j , grid * sizeof(double), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(beta_k_device , beta_k , grid * sizeof(double), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(lambda_device , lambda , grid * sizeof(double), hipMemcpyHostToDevice));
    cudaCheck(hipGetLastError());

    // Printing out some device data
    struct hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, 0);
    int maxGridSize = properties.maxGridSize[0];
    int maxBlockSize = properties.maxThreadsDim[0];
    int maxThreadCount = properties.maxThreadsPerBlock;
    size_t sharedMemoryPerBlock = properties.sharedMemPerBlock;
    printf("MaxGridDim1 %d, MaxBlockDim1 %d, MaxThreadPerBlock %d, SharedMemPerBlock %d\n", 
            maxGridSize,    maxBlockSize,    maxThreadCount,       sharedMemoryPerBlock);

    // Dimension
    // TODO, need to figure out how many
    long numOfThreads = 64;
    long numOfBlocks = ceil(grid/numOfThreads);

    dim3 dimBlock(numOfThreads);
    dim3 dimGrid(numOfBlocks);

    printf("Config: #ofThreads %d, #ofBlocks %d\n", numOfThreads, numOfBlocks);

    hipEvent_t start, stop;
    float et;
    cudaCheck(hipEventCreate(&start));
    cudaCheck(hipEventCreate(&stop));
    cudaCheck(hipEventRecord(start));

    printf("GSRBCuda Starting..\n");
    auto t1 = std::chrono::high_resolution_clock::now();

    for (int timestep = 0; timestep < 4; timestep++)
    {
      // Cuda Kernel Call
      GSRBKernel<<<dimGrid, dimBlock>>> (phi_device, phi_new_device, rhs_device, alpha_device, beta_i_device, 
                                         beta_j_device , beta_k_device , lambda_device, 0);

      cudaCheck(hipGetLastError());
      hipDeviceSynchronize();
      cudaCheck(hipGetLastError());

      GSRBKernel<<<dimGrid, dimBlock>>> (phi_device, phi_new_device, rhs_device, alpha_device, beta_i_device, 
                                         beta_j_device , beta_k_device , lambda_device, 1);

      cudaCheck(hipGetLastError());
      hipDeviceSynchronize();
      cudaCheck(hipGetLastError());

      tmp = phi_new_device;
      phi_new_device = phi_device;
      phi_device = tmp;
    }

    auto t2 = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> fp_ms = t2 - t1;

    std::cout << "CUDA Time is "
              << fp_ms.count()
              << " milliseconds\n";

    // Time event end
    cudaCheck(hipEventRecord(stop));
    cudaCheck(hipEventSynchronize(stop));
    cudaCheck(hipEventElapsedTime(&et, start, stop));
    cudaCheck(hipEventDestroy(start));
    cudaCheck(hipEventDestroy(stop));

    printf("Cuda Time is %f\n", et);

    // More Memory Management
    cudaCheck(hipMemcpy(phi    , phi_device,     grid * sizeof(double), hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(phi_new, phi_new_device, grid * sizeof(double), hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(rhs    , rhs_device,     grid * sizeof(double), hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(alpha  , alpha_device,   grid * sizeof(double), hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(beta_i , beta_i_device,  grid * sizeof(double), hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(beta_j , beta_j_device,  grid * sizeof(double), hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(beta_k , beta_k_device,  grid * sizeof(double), hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(lambda , lambda_device,  grid * sizeof(double), hipMemcpyDeviceToHost));
    cudaCheck(hipGetLastError());

    hipFree(phi_device    );
    hipFree(phi_new_device);
    hipFree(rhs_device    );
    hipFree(alpha_device  );
    hipFree(beta_i_device );
    hipFree(beta_j_device );
    hipFree(beta_k_device );
    hipFree(lambda_device );
}
