#include "hip/hip_runtime.h"
#include "GSRBCuda.h"

#define pencil 32
#define plane  1024
#define grid   32768

#define DEFAULT_THRESHOLD  4000
#define BLOCKSIZE 64
#define TILESIZE 16
#define cudaCheck(x) _cudaCheck(x, #x ,__FILE__, __LINE__)

template<typename T>
void _cudaCheck(T e, const char* func, const char* call, const int line){
  if(e != hipSuccess){
    printf("\"%s\" at %d in %s\n\treturned %d\n-> %s\n", func, line, call, (int)e, hipGetErrorString(e));
    exit(EXIT_FAILURE);
  }
}

__global__ void GSRBKernel(double* phi, double* phi_new, double* rhs, double* alpha, double* beta_i,
                           double* beta_j, double* beta_k, double* lambda, int color)
{
    // TODO: Find out what i, j, k maps to in terms of blockIdx, blockDim, threadIdx
    int currentOffset = blockIdx.x + blockDim.x + threadIdx.x;

    int i, j, k;
    double h2inv = 1.0/64;
    k = currentOffset;

    for (j=0; j<pencil; j++)
    {
        for(i=0; i<pencil; i++)
        {

    int ijk = i + j*pencil + k*plane;

    if (i+j+k+color % 2 == 0)
    {
        double helmholtz = alpha[ijk]*phi[ijk]
                         - h2inv*(
                               beta_i[ijk+1     ]*( phi[ijk+1     ]-phi[ijk       ] )
                             - beta_i[ijk       ]*( phi[ijk       ]-phi[ijk-1     ] )
                             + beta_j[ijk+pencil]*( phi[ijk+pencil]-phi[ijk       ] )
                             - beta_j[ijk       ]*( phi[ijk       ]-phi[ijk-pencil] )
                             + beta_k[ijk+plane ]*( phi[ijk+plane ]-phi[ijk       ] )
                             - beta_k[ijk       ]*( phi[ijk       ]-phi[ijk-plane ] )
                              );

        phi_new[ijk] = phi[ijk] - lambda[ijk]*(helmholtz-rhs[ijk]);
    }
        }
    }
}

void GSRBCuda(double* phi, double* phi_new, double* rhs, double* alpha, double* beta_i,
              double* beta_j, double* beta_k, double* lambda)
{
    //CUDA Buffers
    double* phi_device    ;
    double* phi_new_device;
    double* rhs_device    ;
    double* alpha_device  ;
    double* beta_i_device ;
    double* beta_j_device ;
    double* beta_k_device ;
    double* lambda_device ;

    // Init Memory on GPU
    // Cuda Memory Management
    cudaCheck(hipMalloc((void**) &phi_device    , grid * sizeof(double)));
    cudaCheck(hipMalloc((void**) &phi_new_device, grid * sizeof(double)));
    cudaCheck(hipMalloc((void**) &rhs_device    , grid * sizeof(double)));
    cudaCheck(hipMalloc((void**) &alpha_device  , grid * sizeof(double)));
    cudaCheck(hipMalloc((void**) &beta_i_device , grid * sizeof(double)));
    cudaCheck(hipMalloc((void**) &beta_j_device , grid * sizeof(double)));
    cudaCheck(hipMalloc((void**) &beta_k_device , grid * sizeof(double)));
    cudaCheck(hipMalloc((void**) &lambda_device , grid * sizeof(double)));
    cudaCheck(hipGetLastError());

    cudaCheck(hipMemcpy(phi_device    , phi    , grid * sizeof(double), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(phi_new_device, phi_new, grid * sizeof(double), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(rhs_device    , rhs    , grid * sizeof(double), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(alpha_device  , alpha  , grid * sizeof(double), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(beta_i_device , beta_i , grid * sizeof(double), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(beta_j_device , beta_j , grid * sizeof(double), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(beta_k_device , beta_k , grid * sizeof(double), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(lambda_device , lambda , grid * sizeof(double), hipMemcpyHostToDevice));
    cudaCheck(hipGetLastError());

    // Printing out some device data
    struct hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, 0);
    int maxGridSize = properties.maxGridSize[0];
    int maxBlockSize = properties.maxThreadsDim[0];
    int maxThreadCount = properties.maxThreadsPerBlock;
    size_t sharedMemoryPerBlock = properties.sharedMemPerBlock;
    printf("MaxGridDim1 %d, MaxBlockDim1 %d, MaxThreadPerBlock %d, SharedMemPerBlock %d\n", 
            maxGridSize,    maxBlockSize,    maxThreadCount,       sharedMemoryPerBlock);

    // Dimension
    // TODO, need to figure out how many
    long numOfBlocks = 50;

    dim3 dimGrid(numOfBlocks);
    dim3 dimBlock(numOfBlocks);

    printf("Config: #ofBlocks %d, #ofThreads %d\n", numOfBlocks, numOfBlocks);

    hipEvent_t start, stop;
    float et;
    cudaCheck(hipEventCreate(&start));
    cudaCheck(hipEventCreate(&stop));
    cudaCheck(hipEventRecord(start));

    // Cuda Kernel Call
    GSRBKernel<<<dimGrid, dimBlock>>> (phi_device, phi_new_device, rhs_device, alpha_device, beta_i_device, 
                                       beta_j_device , beta_k_device , lambda_device, color);

    cudaCheck(hipGetLastError());
    hipDeviceSynchronize();
    cudaCheck(hipGetLastError());

    // Time event end
    cudaCheck(hipEventRecord(stop));
    cudaCheck(hipEventSynchronize(stop));
    cudaCheck(hipEventElapsedTime(&et, start, stop));
    cudaCheck(hipEventDestroy(start));
    cudaCheck(hipEventDestroy(stop));

    printf("Cuda Time is %f\n", et);

    // More Memory Management
    // hipMemcpy(host_output, device_output, inputSize, hipMemcpyDeviceToHost);
    // hipFree(device_output);
    // hipFree(device_input);
}
