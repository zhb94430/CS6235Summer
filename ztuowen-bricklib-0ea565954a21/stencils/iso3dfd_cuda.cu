#include "hip/hip_runtime.h"
// this source is derived from CHILL AST originally from file 'iso3dfd.cpp' as parsed by frontend compiler clang
#include "brick.h"
#include <hip/hip_runtime.h>
#include "brick-cuda.h"
#include "iso3dfd_cuda.h"

namespace {
  __constant__ float coeff[5];
}

__global__ void
__launch_bounds__(BLOCKDIM, NBLOCK)
_cuda_iso3dfd(struct brickd ptr_next, struct brickd ptr_prev, struct brickd ptr_vel, struct brick_list blist) {
  int lid = threadIdx.x & 31;
  int wid = (threadIdx.x >> 5);
  brick_info *binfo = ptr_next.info;
  if (wid < WARP)
    for (int r = blockIdx.x; r < blist.rlen; r += GRIDDIM)
#ifdef CYCL
      for (int o = blist.bdat[blist.rdat[r]] + wid; o < blist.bdat[blist.rdat[r + 1]]; o+=WARP) {
#else
        for (int l = blist.rdat[r] + wid; l < blist.rdat[r + 1]; l += WARP)
          for (int o = blist.bdat[l]; o < blist.bdat[l + 1]; ++o) {
#endif
      __syncthreads();
      long b = blist.dat[o];
      {
      brick_info *binfo = ptr_next.info;
      long b0 = binfo->adj[b][0];
      long b1 = binfo->adj[b][1];
      long b2 = binfo->adj[b][2];
      long b3 = binfo->adj[b][3];
      long b4 = binfo->adj[b][4];
      long b5 = binfo->adj[b][5];
      long b6 = binfo->adj[b][6];
      long b7 = binfo->adj[b][7];
      long b8 = binfo->adj[b][8];
      long b9 = binfo->adj[b][9];
      long b10 = binfo->adj[b][10];
      long b11 = binfo->adj[b][11];
      long b12 = binfo->adj[b][12];
      long b13 = b;
      long b14 = binfo->adj[b][13];
      long b15 = binfo->adj[b][14];
      long b16 = binfo->adj[b][15];
      long b17 = binfo->adj[b][16];
      long b18 = binfo->adj[b][17];
      long b19 = binfo->adj[b][18];
      long b20 = binfo->adj[b][19];
      long b21 = binfo->adj[b][20];
      long b22 = binfo->adj[b][21];
      long b23 = binfo->adj[b][22];
      long b24 = binfo->adj[b][23];
      long b25 = binfo->adj[b][24];
      long b26 = binfo->adj[b][25];
      float buf1[16];
      float buf0[16];
      {
        for (long iit = 0; iit < 16; ++iit)
          buf0[iit] = 0;
        float ptr_prev_0_0_0_l;
        float ptr_prev_0_0_0_r;
        {
          {
            long irel2 = 0;
            for (long zit = 0; zit < 4; zit += 1) {
              long rel = irel2;
              {
                long irel1 = rel;
                for (long yit = 0; yit < 4; yit += 1) {
                  long rel = irel1;
                  // Alignment 28, 4, 4
                  {
                    float ptr_prev_0_0_0;
                    {
                      {
                        ptr_prev_0_0_0_l = ptr_prev.dat[b12 * ptr_prev.step + 0 + rel * 32 + lid];
                      }
                      {
                        ptr_prev_0_0_0_r = ptr_prev.dat[b13 * ptr_prev.step + 0 + rel * 32 + lid];
                      }
                      cu_shl(ptr_prev_0_0_0, ptr_prev_0_0_0_l, ptr_prev_0_0_0_r, 4, 32, lid);
                    }
                    {
                      buf0[rel + 0] += ptr_prev_0_0_0 * coeff[4];
                    }
                  }
                  // Alignment 29, 4, 4
                  {
                    float ptr_prev_0_0_0;
                    {
                      cu_shl(ptr_prev_0_0_0, ptr_prev_0_0_0_l, ptr_prev_0_0_0_r, 3, 32, lid);
                    }
                    {
                      buf0[rel + 0] += ptr_prev_0_0_0 * coeff[3];
                    }
                  }
                  // Alignment 30, 4, 4
                  {
                    float ptr_prev_0_0_0;
                    {
                      cu_shl(ptr_prev_0_0_0, ptr_prev_0_0_0_l, ptr_prev_0_0_0_r, 2, 32, lid);
                    }
                    {
                      buf0[rel + 0] += ptr_prev_0_0_0 * coeff[2];
                    }
                  }
                  // Alignment 31, 4, 4
                  {
                    float ptr_prev_0_0_0;
                    {
                      cu_shl(ptr_prev_0_0_0, ptr_prev_0_0_0_l, ptr_prev_0_0_0_r, 1, 32, lid);
                    }
                    {
                      buf0[rel + 0] += ptr_prev_0_0_0 * coeff[1];
                    }
                  }
                  // Alignment 32, 4, 4
                  {
                    float ptr_prev_0_0_0;
                    {
                      ptr_prev_0_0_0_l = ptr_prev_0_0_0_r;
                      ptr_prev_0_0_0 = ptr_prev_0_0_0_l;
                    }
                    {
                      buf0[rel + 0] += ptr_prev_0_0_0 * coeff[0];
                    }
                  }
                  // Alignment 33, 4, 4
                  {
                    float ptr_prev_0_0_0;
                    {
                      {
                        ptr_prev_0_0_0_r = ptr_prev.dat[b14 * ptr_prev.step + 0 + rel * 32 + lid];
                      }
                      cu_shl(ptr_prev_0_0_0, ptr_prev_0_0_0_l, ptr_prev_0_0_0_r, 31, 32, lid);
                    }
                    {
                      buf0[rel + 0] += ptr_prev_0_0_0 * coeff[1];
                    }
                  }
                  // Alignment 34, 4, 4
                  {
                    float ptr_prev_0_0_0;
                    {
                      cu_shl(ptr_prev_0_0_0, ptr_prev_0_0_0_l, ptr_prev_0_0_0_r, 30, 32, lid);
                    }
                    {
                      buf0[rel + 0] += ptr_prev_0_0_0 * coeff[2];
                    }
                  }
                  // Alignment 35, 4, 4
                  {
                    float ptr_prev_0_0_0;
                    {
                      cu_shl(ptr_prev_0_0_0, ptr_prev_0_0_0_l, ptr_prev_0_0_0_r, 29, 32, lid);
                    }
                    {
                      buf0[rel + 0] += ptr_prev_0_0_0 * coeff[3];
                    }
                  }
                  // Alignment 36, 4, 4
                  {
                    float ptr_prev_0_0_0;
                    {
                      cu_shl(ptr_prev_0_0_0, ptr_prev_0_0_0_l, ptr_prev_0_0_0_r, 28, 32, lid);
                    }
                    {
                      buf0[rel + 0] += ptr_prev_0_0_0 * coeff[4];
                    }
                  }
                  irel1 += 1;
                }
              }
              irel2 += 4;
            }
          }
          {
            long irel2 = 0;
            for (long zit = 0; zit < 4; zit += 1) {
              long rel = irel2;
              {
                // Alignment 32, 0, 4
                {
                  float ptr_prev_0_0_0;
                  {
                    {
                      ptr_prev_0_0_0_l = ptr_prev.dat[b10 * ptr_prev.step + 0 + rel * 32 + lid];
                    }
                    ptr_prev_0_0_0 = ptr_prev_0_0_0_l;
                  }
                  {
                    buf0[rel + 0] += ptr_prev_0_0_0 * coeff[4];
                  }
                }
                // Alignment 32, 1, 4
                {
                  float ptr_prev_0_0_0;
                  {
                    {
                      ptr_prev_0_0_0_l = ptr_prev.dat[b10 * ptr_prev.step + 32 + rel * 32 + lid];
                    }
                    ptr_prev_0_0_0 = ptr_prev_0_0_0_l;
                  }
                  {
                    buf0[rel + 0] += ptr_prev_0_0_0 * coeff[3];
                    buf0[rel + 1] += ptr_prev_0_0_0 * coeff[4];
                  }
                }
                // Alignment 32, 2, 4
                {
                  float ptr_prev_0_0_0;
                  {
                    {
                      ptr_prev_0_0_0_l = ptr_prev.dat[b10 * ptr_prev.step + 64 + rel * 32 + lid];
                    }
                    ptr_prev_0_0_0 = ptr_prev_0_0_0_l;
                  }
                  {
                    buf0[rel + 0] += ptr_prev_0_0_0 * coeff[2];
                    buf0[rel + 1] += ptr_prev_0_0_0 * coeff[3];
                    buf0[rel + 2] += ptr_prev_0_0_0 * coeff[4];
                  }
                }
                // Alignment 32, 3, 4
                {
                  float ptr_prev_0_0_0;
                  {
                    {
                      ptr_prev_0_0_0_l = ptr_prev.dat[b10 * ptr_prev.step + 96 + rel * 32 + lid];
                    }
                    ptr_prev_0_0_0 = ptr_prev_0_0_0_l;
                  }
                  {
                    buf0[rel + 0] += ptr_prev_0_0_0 * coeff[1];
                    buf0[rel + 1] += ptr_prev_0_0_0 * coeff[2];
                    buf0[rel + 2] += ptr_prev_0_0_0 * coeff[3];
                    buf0[rel + 3] += ptr_prev_0_0_0 * coeff[4];
                  }
                }
                // Alignment 32, 4, 4
                {
                  float ptr_prev_0_0_0;
                  {
                    {
                      ptr_prev_0_0_0_l = ptr_prev.dat[b13 * ptr_prev.step + 0 + rel * 32 + lid];
                    }
                    ptr_prev_0_0_0 = ptr_prev_0_0_0_l;
                  }
                  {
                    buf0[rel + 1] += ptr_prev_0_0_0 * coeff[1];
                    buf0[rel + 2] += ptr_prev_0_0_0 * coeff[2];
                    buf0[rel + 3] += ptr_prev_0_0_0 * coeff[3];
                  }
                }
                // Alignment 32, 5, 4
                {
                  float ptr_prev_0_0_0;
                  {
                    {
                      ptr_prev_0_0_0_l = ptr_prev.dat[b13 * ptr_prev.step + 32 + rel * 32 + lid];
                    }
                    ptr_prev_0_0_0 = ptr_prev_0_0_0_l;
                  }
                  {
                    buf0[rel + 0] += ptr_prev_0_0_0 * coeff[1];
                    buf0[rel + 2] += ptr_prev_0_0_0 * coeff[1];
                    buf0[rel + 3] += ptr_prev_0_0_0 * coeff[2];
                  }
                }
                // Alignment 32, 6, 4
                {
                  float ptr_prev_0_0_0;
                  {
                    {
                      ptr_prev_0_0_0_l = ptr_prev.dat[b13 * ptr_prev.step + 64 + rel * 32 + lid];
                    }
                    ptr_prev_0_0_0 = ptr_prev_0_0_0_l;
                  }
                  {
                    buf0[rel + 1] += ptr_prev_0_0_0 * coeff[1];
                    buf0[rel + 3] += ptr_prev_0_0_0 * coeff[1];
                    buf0[rel + 0] += ptr_prev_0_0_0 * coeff[2];
                  }
                }
                // Alignment 32, 7, 4
                {
                  float ptr_prev_0_0_0;
                  {
                    {
                      ptr_prev_0_0_0_l = ptr_prev.dat[b13 * ptr_prev.step + 96 + rel * 32 + lid];
                    }
                    ptr_prev_0_0_0 = ptr_prev_0_0_0_l;
                  }
                  {
                    buf0[rel + 2] += ptr_prev_0_0_0 * coeff[1];
                    buf0[rel + 1] += ptr_prev_0_0_0 * coeff[2];
                    buf0[rel + 0] += ptr_prev_0_0_0 * coeff[3];
                  }
                }
                // Alignment 32, 8, 4
                {
                  float ptr_prev_0_0_0;
                  {
                    {
                      ptr_prev_0_0_0_l = ptr_prev.dat[b16 * ptr_prev.step + 0 + rel * 32 + lid];
                    }
                    ptr_prev_0_0_0 = ptr_prev_0_0_0_l;
                  }
                  {
                    buf0[rel + 3] += ptr_prev_0_0_0 * coeff[1];
                    buf0[rel + 2] += ptr_prev_0_0_0 * coeff[2];
                    buf0[rel + 1] += ptr_prev_0_0_0 * coeff[3];
                    buf0[rel + 0] += ptr_prev_0_0_0 * coeff[4];
                  }
                }
                // Alignment 32, 9, 4
                {
                  float ptr_prev_0_0_0;
                  {
                    {
                      ptr_prev_0_0_0_l = ptr_prev.dat[b16 * ptr_prev.step + 32 + rel * 32 + lid];
                    }
                    ptr_prev_0_0_0 = ptr_prev_0_0_0_l;
                  }
                  {
                    buf0[rel + 3] += ptr_prev_0_0_0 * coeff[2];
                    buf0[rel + 2] += ptr_prev_0_0_0 * coeff[3];
                    buf0[rel + 1] += ptr_prev_0_0_0 * coeff[4];
                  }
                }
                // Alignment 32, 10, 4
                {
                  float ptr_prev_0_0_0;
                  {
                    {
                      ptr_prev_0_0_0_l = ptr_prev.dat[b16 * ptr_prev.step + 64 + rel * 32 + lid];
                    }
                    ptr_prev_0_0_0 = ptr_prev_0_0_0_l;
                  }
                  {
                    buf0[rel + 3] += ptr_prev_0_0_0 * coeff[3];
                    buf0[rel + 2] += ptr_prev_0_0_0 * coeff[4];
                  }
                }
                // Alignment 32, 11, 4
                {
                  float ptr_prev_0_0_0;
                  {
                    {
                      ptr_prev_0_0_0_l = ptr_prev.dat[b16 * ptr_prev.step + 96 + rel * 32 + lid];
                    }
                    ptr_prev_0_0_0 = ptr_prev_0_0_0_l;
                  }
                  {
                    buf0[rel + 3] += ptr_prev_0_0_0 * coeff[4];
                  }
                }
              }
              irel2 += 4;
            }
          }
          {
            long irel1 = 0;
            for (long yit = 0; yit < 4; yit += 1) {
              long rel = irel1;
              {
                // Alignment 32, 4, 0
                {
                  float ptr_prev_0_0_0;
                  {
                    {
                      ptr_prev_0_0_0_l = ptr_prev.dat[b4 * ptr_prev.step + 0 + rel * 32 + lid];
                    }
                    ptr_prev_0_0_0 = ptr_prev_0_0_0_l;
                  }
                  {
                    buf0[rel + 0] += ptr_prev_0_0_0 * coeff[4];
                  }
                }
                // Alignment 32, 4, 1
                {
                  float ptr_prev_0_0_0;
                  {
                    {
                      ptr_prev_0_0_0_l = ptr_prev.dat[b4 * ptr_prev.step + 128 + rel * 32 + lid];
                    }
                    ptr_prev_0_0_0 = ptr_prev_0_0_0_l;
                  }
                  {
                    buf0[rel + 0] += ptr_prev_0_0_0 * coeff[3];
                    buf0[rel + 4] += ptr_prev_0_0_0 * coeff[4];
                  }
                }
                // Alignment 32, 4, 2
                {
                  float ptr_prev_0_0_0;
                  {
                    {
                      ptr_prev_0_0_0_l = ptr_prev.dat[b4 * ptr_prev.step + 256 + rel * 32 + lid];
                    }
                    ptr_prev_0_0_0 = ptr_prev_0_0_0_l;
                  }
                  {
                    buf0[rel + 0] += ptr_prev_0_0_0 * coeff[2];
                    buf0[rel + 4] += ptr_prev_0_0_0 * coeff[3];
                    buf0[rel + 8] += ptr_prev_0_0_0 * coeff[4];
                  }
                }
                // Alignment 32, 4, 3
                {
                  float ptr_prev_0_0_0;
                  {
                    {
                      ptr_prev_0_0_0_l = ptr_prev.dat[b4 * ptr_prev.step + 384 + rel * 32 + lid];
                    }
                    ptr_prev_0_0_0 = ptr_prev_0_0_0_l;
                  }
                  {
                    buf0[rel + 0] += ptr_prev_0_0_0 * coeff[1];
                    buf0[rel + 4] += ptr_prev_0_0_0 * coeff[2];
                    buf0[rel + 8] += ptr_prev_0_0_0 * coeff[3];
                    buf0[rel + 12] += ptr_prev_0_0_0 * coeff[4];
                  }
                }
                // Alignment 32, 4, 4
                {
                  float ptr_prev_0_0_0;
                  {
                    {
                      ptr_prev_0_0_0_l = ptr_prev.dat[b13 * ptr_prev.step + 0 + rel * 32 + lid];
                    }
                    ptr_prev_0_0_0 = ptr_prev_0_0_0_l;
                  }
                  {
                    buf0[rel + 4] += ptr_prev_0_0_0 * coeff[1];
                    buf0[rel + 8] += ptr_prev_0_0_0 * coeff[2];
                    buf0[rel + 12] += ptr_prev_0_0_0 * coeff[3];
                  }
                }
                // Alignment 32, 4, 5
                {
                  float ptr_prev_0_0_0;
                  {
                    {
                      ptr_prev_0_0_0_l = ptr_prev.dat[b13 * ptr_prev.step + 128 + rel * 32 + lid];
                    }
                    ptr_prev_0_0_0 = ptr_prev_0_0_0_l;
                  }
                  {
                    buf0[rel + 0] += ptr_prev_0_0_0 * coeff[1];
                    buf0[rel + 8] += ptr_prev_0_0_0 * coeff[1];
                    buf0[rel + 12] += ptr_prev_0_0_0 * coeff[2];
                  }
                }
                // Alignment 32, 4, 6
                {
                  float ptr_prev_0_0_0;
                  {
                    {
                      ptr_prev_0_0_0_l = ptr_prev.dat[b13 * ptr_prev.step + 256 + rel * 32 + lid];
                    }
                    ptr_prev_0_0_0 = ptr_prev_0_0_0_l;
                  }
                  {
                    buf0[rel + 4] += ptr_prev_0_0_0 * coeff[1];
                    buf0[rel + 12] += ptr_prev_0_0_0 * coeff[1];
                    buf0[rel + 0] += ptr_prev_0_0_0 * coeff[2];
                  }
                }
                // Alignment 32, 4, 7
                {
                  float ptr_prev_0_0_0;
                  {
                    {
                      ptr_prev_0_0_0_l = ptr_prev.dat[b13 * ptr_prev.step + 384 + rel * 32 + lid];
                    }
                    ptr_prev_0_0_0 = ptr_prev_0_0_0_l;
                  }
                  {
                    buf0[rel + 8] += ptr_prev_0_0_0 * coeff[1];
                    buf0[rel + 4] += ptr_prev_0_0_0 * coeff[2];
                    buf0[rel + 0] += ptr_prev_0_0_0 * coeff[3];
                  }
                }
                // Alignment 32, 4, 8
                {
                  float ptr_prev_0_0_0;
                  {
                    {
                      ptr_prev_0_0_0_l = ptr_prev.dat[b22 * ptr_prev.step + 0 + rel * 32 + lid];
                    }
                    ptr_prev_0_0_0 = ptr_prev_0_0_0_l;
                  }
                  {
                    buf0[rel + 12] += ptr_prev_0_0_0 * coeff[1];
                    buf0[rel + 8] += ptr_prev_0_0_0 * coeff[2];
                    buf0[rel + 4] += ptr_prev_0_0_0 * coeff[3];
                    buf0[rel + 0] += ptr_prev_0_0_0 * coeff[4];
                  }
                }
                // Alignment 32, 4, 9
                {
                  float ptr_prev_0_0_0;
                  {
                    {
                      ptr_prev_0_0_0_l = ptr_prev.dat[b22 * ptr_prev.step + 128 + rel * 32 + lid];
                    }
                    ptr_prev_0_0_0 = ptr_prev_0_0_0_l;
                  }
                  {
                    buf0[rel + 12] += ptr_prev_0_0_0 * coeff[2];
                    buf0[rel + 8] += ptr_prev_0_0_0 * coeff[3];
                    buf0[rel + 4] += ptr_prev_0_0_0 * coeff[4];
                  }
                }
                // Alignment 32, 4, 10
                {
                  float ptr_prev_0_0_0;
                  {
                    {
                      ptr_prev_0_0_0_l = ptr_prev.dat[b22 * ptr_prev.step + 256 + rel * 32 + lid];
                    }
                    ptr_prev_0_0_0 = ptr_prev_0_0_0_l;
                  }
                  {
                    buf0[rel + 12] += ptr_prev_0_0_0 * coeff[3];
                    buf0[rel + 8] += ptr_prev_0_0_0 * coeff[4];
                  }
                }
                // Alignment 32, 4, 11
                {
                  float ptr_prev_0_0_0;
                  {
                    {
                      ptr_prev_0_0_0_l = ptr_prev.dat[b22 * ptr_prev.step + 384 + rel * 32 + lid];
                    }
                    ptr_prev_0_0_0 = ptr_prev_0_0_0_l;
                  }
                  {
                    buf0[rel + 12] += ptr_prev_0_0_0 * coeff[4];
                  }
                }
              }
              irel1 += 1;
            }
          }
        }
      }
      {
        for (long iit = 0; iit < 16; ++iit)
          buf1[iit] = 0;
        float ptr_next_0_0_0_l;
        float ptr_next_0_0_0_r;
        float ptr_prev_0_0_0_l;
        float ptr_prev_0_0_0_r;
        float ptr_vel_0_0_0_l;
        float ptr_vel_0_0_0_r;
        {
          {
            long irel2 = 0;
            for (long zit = 0; zit < 4; zit += 1) {
              long rel = irel2;
              {
                long irel1 = rel;
                for (long yit = 0; yit < 4; yit += 1) {
                  long rel = irel1;
                  // Alignment 32, 4, 4
                  {
                    float ptr_prev_0_0_0;
                    {
                      {
                        ptr_prev_0_0_0_l = ptr_prev.dat[b13 * ptr_prev.step + 0 + rel * 32 + lid];
                      }
                      ptr_prev_0_0_0 = ptr_prev_0_0_0_l;
                    }
                    float ptr_vel_0_0_0;
                    {
                      {
                        ptr_vel_0_0_0_l = ptr_vel.dat[b13 * ptr_vel.step + 0 + rel * 32 + lid];
                      }
                      ptr_vel_0_0_0 = ptr_vel_0_0_0_l;
                    }
                    float ptr_next_0_0_0;
                    {
                      {
                        ptr_next_0_0_0_l = ptr_next.dat[b13 * ptr_next.step + 0 + rel * 32 + lid];
                      }
                      ptr_next_0_0_0 = ptr_next_0_0_0_l;
                    }
                    {
                      buf1[rel + 0] += 2.0f * ptr_prev_0_0_0;
                      buf1[rel + 0] += buf0[0 + rel] * ptr_vel_0_0_0;
                      buf1[rel + 0] += (-ptr_next_0_0_0);
                    }
                  }
                  irel1 += 1;
                }
              }
              irel2 += 4;
            }
          }
        }
      }
      {
        float *stp = &ptr_next.dat[ptr_next.step * b];
        for (long sti = 0; sti < 16; ++sti)
          stp[sti * 32 + lid] = buf1[sti];
      }
    }
  }
}

int cuda_iso3dfd(struct brickd &ptr_next, struct brickd &ptr_prev, struct brickd &ptr_vel, struct brick_list &blist) {
  dim3 grid(GRIDDIM), block(BLOCKDIM);
  _cuda_iso3dfd<<<grid,block>>>(ptr_next, ptr_prev, ptr_vel, blist);
  cudaCheck(hipGetLastError());
  return 1;
}

void cuda_iso3dfd_setup(float *coeff_host) {
  hipMemcpyToSymbol(HIP_SYMBOL(coeff), coeff_host, sizeof(float)*5);
}
