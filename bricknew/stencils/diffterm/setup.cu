#include "hip/hip_runtime.h"
#include "brick-cuda.h"
#include "head.h"
#include "headcu.h"

#define q_1(i, j, k) q_1_arr[k][j][i]
#define q_2(i, j, k) q_2_arr[k][j][i]
#define q_3(i, j, k) q_3_arr[k][j][i]
#define q_5(i, j, k) q_5_arr[k][j][i]

#define ux(i, j, k) ux_arr[k][j][i]
#define vx(i, j, k) vx_arr[k][j][i]
#define wx(i, j, k) wx_arr[k][j][i]

#define uy(i, j, k) uy_arr[k][j][i]
#define vy(i, j, k) vy_arr[k][j][i]
#define wy(i, j, k) wy_arr[k][j][i]

#define uz(i, j, k) uz_arr[k][j][i]
#define vz(i, j, k) vz_arr[k][j][i]
#define wz(i, j, k) wz_arr[k][j][i]

#define diffflux1(i, j, k) diffflux1_arr[k][j][i]
#define diffflux2(i, j, k) diffflux2_arr[k][j][i]
#define diffflux3(i, j, k) diffflux3_arr[k][j][i]
#define diffflux4(i, j, k) diffflux4_arr[k][j][i]

#define dxinv0 c[0]
#define dxinv1 c[1]
#define dxinv2 c[2]

__global__ void
arr_kernel1(bElem *q_1_ptr, bElem *q_2_ptr, bElem *q_3_ptr, 
        bElem *ux_ptr, bElem *vx_ptr, bElem *wx_ptr, 
        bElem *uy_ptr, bElem *vy_ptr, bElem *wy_ptr, 
        bElem *uz_ptr, bElem *vz_ptr, bElem *wz_ptr, 
        bElem *c) {
    auto q_1_arr = (bElem (*)[STRIDE][STRIDE]) q_1_ptr;
    auto q_2_arr = (bElem (*)[STRIDE][STRIDE]) q_2_ptr;
    auto q_3_arr = (bElem (*)[STRIDE][STRIDE]) q_3_ptr;

    auto ux_arr = (bElem (*)[STRIDE][STRIDE]) ux_ptr;
    auto vx_arr = (bElem (*)[STRIDE][STRIDE]) vx_ptr;
    auto wx_arr = (bElem (*)[STRIDE][STRIDE]) wx_ptr;

    auto uy_arr = (bElem (*)[STRIDE][STRIDE]) uy_ptr;
    auto vy_arr = (bElem (*)[STRIDE][STRIDE]) vy_ptr;
    auto wy_arr = (bElem (*)[STRIDE][STRIDE]) wy_ptr;

    auto uz_arr = (bElem (*)[STRIDE][STRIDE]) uz_ptr;
    auto vz_arr = (bElem (*)[STRIDE][STRIDE]) vz_ptr;
    auto wz_arr = (bElem (*)[STRIDE][STRIDE]) wz_ptr;

    #include "arrcusched.h"
    {
        #include "kernel1.h"
    }
}

__global__ void
arr_kernel2(bElem *q_1_ptr, 
        bElem *vy_ptr, 
        bElem *wz_ptr, 
        bElem *diffflux1_ptr,
        bElem *c) {
    auto q_1_arr = (bElem (*)[STRIDE][STRIDE]) q_1_ptr;

    auto vy_arr = (bElem (*)[STRIDE][STRIDE]) vy_ptr;
    auto wz_arr = (bElem (*)[STRIDE][STRIDE]) wz_ptr;

    auto diffflux1_arr = (bElem (*)[STRIDE][STRIDE]) diffflux1_ptr;

    #include "arrcusched.h"
    {
        #include "kernel2.h"
    }
}

__global__ void
arr_kernel3(bElem *q_2_ptr, 
        bElem *ux_ptr, bElem *wz_ptr, 
        bElem *diffflux2_ptr,
        bElem *c) {
    auto q_2_arr = (bElem (*)[STRIDE][STRIDE]) q_2_ptr;

    auto ux_arr = (bElem (*)[STRIDE][STRIDE]) ux_ptr;
    auto wz_arr = (bElem (*)[STRIDE][STRIDE]) wz_ptr;

    auto diffflux2_arr = (bElem (*)[STRIDE][STRIDE]) diffflux2_ptr;

    #include "arrcusched.h"
    {
        #include "kernel3.h"
    }
}

__global__ void
arr_kernel4(bElem *q_3_ptr, 
        bElem *ux_ptr, bElem *vy_ptr, 
        bElem *diffflux3_ptr,
        bElem *c) {
    auto q_3_arr = (bElem (*)[STRIDE][STRIDE]) q_3_ptr;

    auto ux_arr = (bElem (*)[STRIDE][STRIDE]) ux_ptr;
    auto vy_arr = (bElem (*)[STRIDE][STRIDE]) vy_ptr;

    auto diffflux3_arr = (bElem (*)[STRIDE][STRIDE]) diffflux3_ptr;

    #include "arrcusched.h"
    {
        #include "kernel4.h"
    }
}

__global__ void
arr_kernel5(bElem *q_1_ptr, bElem *q_2_ptr, bElem *q_3_ptr, bElem *q_5_ptr,
        bElem *ux_ptr, bElem *vx_ptr, bElem *wx_ptr, 
        bElem *uy_ptr, bElem *vy_ptr, bElem *wy_ptr, 
        bElem *uz_ptr, bElem *vz_ptr, bElem *wz_ptr, 
        bElem *diffflux1_ptr, bElem *diffflux2_ptr, bElem *diffflux3_ptr, bElem *diffflux4_ptr,
        bElem *c) {
    auto q_1_arr = (bElem (*)[STRIDE][STRIDE]) q_1_ptr;
    auto q_2_arr = (bElem (*)[STRIDE][STRIDE]) q_2_ptr;
    auto q_3_arr = (bElem (*)[STRIDE][STRIDE]) q_3_ptr;
    auto q_5_arr = (bElem (*)[STRIDE][STRIDE]) q_5_ptr;

    auto ux_arr = (bElem (*)[STRIDE][STRIDE]) ux_ptr;
    auto vx_arr = (bElem (*)[STRIDE][STRIDE]) vx_ptr;
    auto wx_arr = (bElem (*)[STRIDE][STRIDE]) wx_ptr;

    auto uy_arr = (bElem (*)[STRIDE][STRIDE]) uy_ptr;
    auto vy_arr = (bElem (*)[STRIDE][STRIDE]) vy_ptr;
    auto wy_arr = (bElem (*)[STRIDE][STRIDE]) wy_ptr;

    auto uz_arr = (bElem (*)[STRIDE][STRIDE]) uz_ptr;
    auto vz_arr = (bElem (*)[STRIDE][STRIDE]) vz_ptr;
    auto wz_arr = (bElem (*)[STRIDE][STRIDE]) wz_ptr;

    auto diffflux1_arr = (bElem (*)[STRIDE][STRIDE]) diffflux1_ptr;
    auto diffflux2_arr = (bElem (*)[STRIDE][STRIDE]) diffflux2_ptr;
    auto diffflux3_arr = (bElem (*)[STRIDE][STRIDE]) diffflux3_ptr;
    auto diffflux4_arr = (bElem (*)[STRIDE][STRIDE]) diffflux4_ptr;

    #include "arrcusched.h"
    {
        #include "kernel5.h"
    }
}

#undef q_1
#undef q_2
#undef q_3
#undef q_5

#undef ux
#undef vx
#undef wx

#undef uy
#undef vy
#undef wy

#undef uz
#undef vz
#undef wz

#undef diffflux1
#undef diffflux2
#undef diffflux3
#undef diffflux4

__global__ void
brick_kernel1(unsigned (*grid)[STRIDE/TILEJ][STRIDE/TILEI], 
        Brick3D q_1, Brick3D q_2, Brick3D q_3,
        Brick3D ux, Brick3D vx, Brick3D wx, 
        Brick3D uy, Brick3D vy, Brick3D wy, 
        Brick3D uz, Brick3D vz, Brick3D wz, bElem *c) {

    #include "bricusched.h"
    brick("kernel1.py", BVEC, (TILEK, TILEJ, TILEI), (BFOLD), b);
}

__global__ void
brick_kernel2(unsigned (*grid)[STRIDE/TILEJ][STRIDE/TILEI], 
        Brick3D q_1, Brick3D vy, Brick3D wz, Brick3D diffflux1, bElem *c) {

    #include "bricusched.h"
    brick("kernel2.py", BVEC, (TILEK, TILEJ, TILEI), (BFOLD), b);
}

__global__ void
brick_kernel3(unsigned (*grid)[STRIDE/TILEJ][STRIDE/TILEI], 
        Brick3D q_2, Brick3D ux, Brick3D wz, Brick3D diffflux2, bElem *c) {

    #include "bricusched.h"
    brick("kernel3.py", BVEC, (TILEK, TILEJ, TILEI), (BFOLD), b);
}

__global__ void
brick_kernel4(unsigned (*grid)[STRIDE/TILEJ][STRIDE/TILEI], 
        Brick3D q_3, Brick3D ux, Brick3D vy, Brick3D diffflux3, bElem *c) {

    #include "bricusched.h"
    brick("kernel4.py", BVEC, (TILEK, TILEJ, TILEI), (BFOLD), b);
}

__global__ void
brick_kernel5(unsigned (*grid)[STRIDE/TILEJ][STRIDE/TILEI], 
        Brick3D q_1, Brick3D q_2, Brick3D q_3, Brick3D q_5, 
        Brick3D ux, Brick3D vx, Brick3D wx, 
        Brick3D uy, Brick3D vy, Brick3D wy, 
        Brick3D uz, Brick3D vz, Brick3D wz, 
        Brick3D diffflux1, Brick3D diffflux2, Brick3D diffflux3, Brick3D diffflux4, bElem *c) {

    #include "bricusched.h"
    brick("kernel5.py", BVEC, (TILEK, TILEJ, TILEI), (BFOLD), b);
}

int main() {
    // allocations
    bElem *c = randomArray({3});
    bElem *c_dev;
    copyToDevice({3}, c_dev, c);

    auto q_1_arr = randomArray({STRIDE, STRIDE, STRIDE});
    bElem *q_1_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, q_1_dev, q_1_arr);

    auto q_2_arr = randomArray({STRIDE, STRIDE, STRIDE});
    bElem *q_2_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, q_2_dev, q_2_arr);

    auto q_3_arr = randomArray({STRIDE, STRIDE, STRIDE});
    bElem *q_3_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, q_3_dev, q_3_arr);

    auto q_5_arr = randomArray({STRIDE, STRIDE, STRIDE});
    bElem *q_5_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, q_5_dev, q_5_arr);

    auto ux_arr = zeroArray({STRIDE, STRIDE, STRIDE});
    bElem *ux_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, ux_dev, ux_arr);

    auto vx_arr = zeroArray({STRIDE, STRIDE, STRIDE});
    bElem *vx_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, vx_dev, vx_arr);

    auto wx_arr = zeroArray({STRIDE, STRIDE, STRIDE});
    bElem *wx_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, wx_dev, wx_arr);

    auto uy_arr = zeroArray({STRIDE, STRIDE, STRIDE});
    bElem *uy_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, uy_dev, uy_arr);

    auto vy_arr = zeroArray({STRIDE, STRIDE, STRIDE});
    bElem *vy_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, vy_dev, vy_arr);

    auto wy_arr = zeroArray({STRIDE, STRIDE, STRIDE});
    bElem *wy_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, wy_dev, wy_arr);

    auto uz_arr = zeroArray({STRIDE, STRIDE, STRIDE});
    bElem *uz_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, uz_dev, uz_arr);

    auto vz_arr = zeroArray({STRIDE, STRIDE, STRIDE});
    bElem *vz_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, vz_dev, vz_arr);

    auto wz_arr = zeroArray({STRIDE, STRIDE, STRIDE});
    bElem *wz_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, wz_dev, wz_arr);

    auto diffflux1_arr = zeroArray({STRIDE, STRIDE, STRIDE});
    bElem *diffflux1_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, diffflux1_dev, diffflux1_arr);

    auto diffflux2_arr = zeroArray({STRIDE, STRIDE, STRIDE});
    bElem *diffflux2_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, diffflux2_dev, diffflux2_arr);

    auto diffflux3_arr = zeroArray({STRIDE, STRIDE, STRIDE});
    bElem *diffflux3_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, diffflux3_dev, diffflux3_arr);

    auto diffflux4_arr = zeroArray({STRIDE, STRIDE, STRIDE});
    bElem *diffflux4_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, diffflux4_dev, diffflux4_arr);

    {
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(arr_kernel1), hipFuncCachePreferL1);

        hipFuncSetCacheConfig(reinterpret_cast<const void*>(arr_kernel2), hipFuncCachePreferL1);

        hipFuncSetCacheConfig(reinterpret_cast<const void*>(arr_kernel3), hipFuncCachePreferL1);

        hipFuncSetCacheConfig(reinterpret_cast<const void*>(arr_kernel4), hipFuncCachePreferL1);

        hipFuncSetCacheConfig(reinterpret_cast<const void*>(arr_kernel5), hipFuncCachePreferL1);

        auto compute = [&]() -> void {
            dim3 block(N/TILEI, N/TILEJ, N/TILEK), thread(_TILEI, _TILEJ, _TILEK);
            
            arr_kernel1<<< block, thread >>>(q_1_dev, q_2_dev, q_3_dev, ux_dev, vx_dev, wx_dev, uy_dev, vy_dev, wy_dev, uz_dev, vz_dev, wz_dev, c_dev);

            arr_kernel2<<< block, thread >>>(q_1_dev, vy_dev, wz_dev, diffflux1_dev, c_dev);

            arr_kernel3<<< block, thread >>>(q_2_dev, ux_dev, wz_dev, diffflux2_dev, c_dev);

            arr_kernel4<<< block, thread >>>(q_3_dev, ux_dev, vy_dev, diffflux3_dev, c_dev);

            arr_kernel5<<< block, thread >>>(q_1_dev, q_2_dev, q_3_dev, q_5_dev, ux_dev, vx_dev, wx_dev, uy_dev, vy_dev, wy_dev,
                    uz_dev, vz_dev, wz_dev, diffflux1_dev, diffflux2_dev, diffflux3_dev, diffflux4_dev, c_dev);
        };
        
#ifndef TYPE
        #include "cutiming.h"
#else
        compute();
        copyFromDevice({STRIDE, STRIDE, STRIDE}, diffflux4_arr, diffflux4_dev);
#endif
    }

    hipDeviceSynchronize();

    hipFree(q_1_dev);
    hipFree(q_2_dev);
    hipFree(q_3_dev);
    hipFree(q_5_dev);

    hipFree(ux_dev);
    hipFree(vx_dev);
    hipFree(wx_dev);

    hipFree(uy_dev);
    hipFree(vy_dev);
    hipFree(wy_dev);

    hipFree(uz_dev);
    hipFree(vz_dev);
    hipFree(wz_dev);

    hipFree(diffflux1_dev);
    hipFree(diffflux2_dev);
    hipFree(diffflux3_dev);
    hipFree(diffflux4_dev);

#if TYPE == 1
    {
        unsigned *grid_ptr;
        unsigned bSize = TILEK * TILEJ * TILEI;

        auto bInfo = init_grid<3>(grid_ptr, {STRIDE/TILEK, STRIDE/TILEJ, STRIDE/TILEI});

        unsigned *grid_dev;
        copyToDevice({STRIDE/TILEK, STRIDE/TILEJ, STRIDE/TILEI}, grid_dev, grid_ptr);

        BrickInfo<3> *bInfo_dev;
        auto _bInfo_dev = movBrickInfo(bInfo, hipMemcpyHostToDevice);
        {
            unsigned size = sizeof(BrickInfo<3>);
            hipMalloc(&bInfo_dev, size);
            hipMemcpy(bInfo_dev, &_bInfo_dev, size, hipMemcpyHostToDevice);
        }

        auto xStorage = BrickStorage::allocate(bInfo.nbricks, bSize * 3);
        auto yStorage = BrickStorage::allocate(bInfo.nbricks, bSize * 3);
        auto zStorage = BrickStorage::allocate(bInfo.nbricks, bSize * 3);

        auto qStorage = BrickStorage::allocate(bInfo.nbricks, bSize * 4);

        BrickStorage *xStorage_dev;
        BrickStorage _xStorage_dev = movBrickStorage(xStorage, hipMemcpyHostToDevice);
        {
            unsigned size = sizeof(BrickStorage);
            hipMalloc(&xStorage_dev, size);
            hipMemcpy(xStorage_dev, &_xStorage_dev, size, hipMemcpyHostToDevice);
        }

        BrickStorage *yStorage_dev;
        BrickStorage _yStorage_dev = movBrickStorage(yStorage, hipMemcpyHostToDevice);
        {
            unsigned size = sizeof(BrickStorage);
            hipMalloc(&yStorage_dev, size);
            hipMemcpy(yStorage_dev, &_yStorage_dev, size, hipMemcpyHostToDevice);
        }

        BrickStorage *zStorage_dev;
        BrickStorage _zStorage_dev = movBrickStorage(zStorage, hipMemcpyHostToDevice);
        {
            unsigned size = sizeof(BrickStorage);
            hipMalloc(&zStorage_dev, size);
            hipMemcpy(zStorage_dev, &_zStorage_dev, size, hipMemcpyHostToDevice);
        }

        auto difffluxStorage = BrickStorage::allocate(bInfo.nbricks, bSize * 4);

        BrickStorage *difffluxStorage_dev;
        BrickStorage _difffluxStorage_dev = movBrickStorage(difffluxStorage, hipMemcpyHostToDevice);
        {
            unsigned size = sizeof(BrickStorage);
            hipMalloc(&difffluxStorage_dev, size);
            hipMemcpy(difffluxStorage_dev, &_difffluxStorage_dev, size, hipMemcpyHostToDevice);
        }

        Brick3D ux_bri(&bInfo, &xStorage, 0);
        Brick3D vx_bri(&bInfo, &xStorage, bSize);
        Brick3D wx_bri(&bInfo, &xStorage, bSize * 2);

        Brick3D uy_bri(&bInfo, &yStorage, 0);
        Brick3D vy_bri(&bInfo, &yStorage, bSize);
        Brick3D wy_bri(&bInfo, &yStorage, bSize * 2);

        Brick3D uz_bri(&bInfo, &zStorage, 0);
        Brick3D vz_bri(&bInfo, &zStorage, bSize);
        Brick3D wz_bri(&bInfo, &zStorage, bSize * 2);

        Brick3D q1_bri(&bInfo, &qStorage, 0);
        Brick3D q2_bri(&bInfo, &qStorage, bSize);
        Brick3D q3_bri(&bInfo, &qStorage, bSize * 2);

        Brick3D q5_bri(&bInfo, &qStorage, bSize * 3);

        Brick3D diffflux1_bri(&bInfo, &difffluxStorage, 0);
        Brick3D diffflux2_bri(&bInfo, &difffluxStorage, bSize);
        Brick3D diffflux3_bri(&bInfo, &difffluxStorage, bSize * 2);
        Brick3D diffflux4_bri(&bInfo, &difffluxStorage, bSize * 3);

        copyBrick<3>({STRIDE, STRIDE, STRIDE}, q_1_arr, grid_ptr, q1_bri);
        copyBrick<3>({STRIDE, STRIDE, STRIDE}, q_2_arr, grid_ptr, q2_bri);
        copyBrick<3>({STRIDE, STRIDE, STRIDE}, q_3_arr, grid_ptr, q3_bri);

        copyBrick<3>({STRIDE, STRIDE, STRIDE}, q_5_arr, grid_ptr, q5_bri);

        BrickStorage *qStorage_dev;
        BrickStorage _qStorage_dev = movBrickStorage(qStorage, hipMemcpyHostToDevice);
        {
            unsigned size = sizeof(BrickStorage);
            hipMalloc(&qStorage_dev, size);
            hipMemcpy(qStorage_dev, &_qStorage_dev, size, hipMemcpyHostToDevice);
        }

        hipFuncSetCacheConfig(reinterpret_cast<const void*>(brick_kernel1), hipFuncCachePreferL1);

        hipFuncSetCacheConfig(reinterpret_cast<const void*>(brick_kernel2), hipFuncCachePreferL1);

        hipFuncSetCacheConfig(reinterpret_cast<const void*>(brick_kernel3), hipFuncCachePreferL1);

        hipFuncSetCacheConfig(reinterpret_cast<const void*>(brick_kernel4), hipFuncCachePreferL1);

        hipFuncSetCacheConfig(reinterpret_cast<const void*>(brick_kernel5), hipFuncCachePreferL1);

        auto compute = [&]() -> void {

            Brick3D ux(bInfo_dev, &_xStorage_dev, 0);
            Brick3D vx(bInfo_dev, &_xStorage_dev, bSize);
            Brick3D wx(bInfo_dev, &_xStorage_dev, bSize * 2);

            Brick3D uy(bInfo_dev, &_yStorage_dev, 0);
            Brick3D vy(bInfo_dev, &_yStorage_dev, bSize);
            Brick3D wy(bInfo_dev, &_yStorage_dev, bSize * 2);

            Brick3D uz(bInfo_dev, &_zStorage_dev, 0);
            Brick3D vz(bInfo_dev, &_zStorage_dev, bSize);
            Brick3D wz(bInfo_dev, &_zStorage_dev, bSize * 2);

            Brick3D q1(bInfo_dev, &_qStorage_dev, 0);
            Brick3D q2(bInfo_dev, &_qStorage_dev, bSize);
            Brick3D q3(bInfo_dev, &_qStorage_dev, bSize * 2);

            Brick3D q5(bInfo_dev, &_qStorage_dev, bSize * 3);

            Brick3D diffflux1(bInfo_dev, &_difffluxStorage_dev, 0);
            Brick3D diffflux2(bInfo_dev, &_difffluxStorage_dev, bSize);
            Brick3D diffflux3(bInfo_dev, &_difffluxStorage_dev, bSize * 2);
            Brick3D diffflux4(bInfo_dev, &_difffluxStorage_dev, bSize * 3);

            ux.bStorage = xStorage_dev;
            vx.bStorage = xStorage_dev;
            wx.bStorage = xStorage_dev;

            uy.bStorage = yStorage_dev;
            vy.bStorage = yStorage_dev;
            wy.bStorage = yStorage_dev;

            uz.bStorage = zStorage_dev;
            vz.bStorage = zStorage_dev;
            wz.bStorage = zStorage_dev;

            q1.bStorage = qStorage_dev;
            q2.bStorage = qStorage_dev;
            q3.bStorage = qStorage_dev;

            q5.bStorage = qStorage_dev;

            diffflux1.bStorage = difffluxStorage_dev;
            diffflux2.bStorage = difffluxStorage_dev;
            diffflux3.bStorage = difffluxStorage_dev;
            diffflux4.bStorage = difffluxStorage_dev;

            auto grid = (unsigned (*)[STRIDE/TILEJ][STRIDE/TILEI]) grid_dev;
            dim3 block(N/TILEI, N/TILEJ, N/TILEK), thread(32);
            
            brick_kernel1<<< block, thread >>>(grid, q1, q2, q3, ux, vx, wx, uy, vy, wy, uz, vz, wz, c_dev);

            brick_kernel2<<< block, thread >>>(grid, q1, vy, wz, diffflux1, c_dev);

            brick_kernel3<<< block, thread >>>(grid, q2, ux, wz, diffflux2, c_dev);

            brick_kernel4<<< block, thread >>>(grid, q3, ux, vy, diffflux3, c_dev);

            brick_kernel5<<< block, thread >>>(grid, q1, q2, q3, q5, ux, vx, wx, uy, vy, wy,
                    uz, vz, wz, diffflux1, diffflux2, diffflux3, diffflux4, c_dev);
        };

        #include "cutiming.h"

        hipDeviceSynchronize();

        hipMemcpy(difffluxStorage.dat, _difffluxStorage_dev.dat, difffluxStorage.chunks * difffluxStorage.step * sizeof(bElem), hipMemcpyDeviceToHost);

        if (!compareBrick<3>({STRIDE, STRIDE, STRIDE}, diffflux4_arr, grid_ptr, diffflux4_bri))
            return 1;
    }
#endif
    return 0;
}
