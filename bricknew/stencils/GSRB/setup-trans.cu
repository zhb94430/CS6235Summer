#include "hip/hip_runtime.h"
#include "brick-cuda.h"
#include "head.h"
#include "headcu.h"

#define H2INV  0.8f

__global__ void GSRBKernel(double* phi, double* phi_new, double* rhs, double* alpha, double* beta_i,
                           double* beta_j, double* beta_k, double* lambda, int color)
{
    int i, j, k;
    i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= STRIDE-GZ || i < GZ)
    {
       return;
    }

    for (k=GZ; k<STRIDE-GZ; k++)
    {
        for(j=GZ; j<STRIDE-GZ; j++)
        {

            int ijk = i + j*STRIDE + k*(STRIDE*STRIDE);

            // if ((i+j+k+color) % 2 == 0)
            {
                double helmholtz = alpha[ijk]*phi[ijk]
                                 - H2INV*(
                                       beta_i[ijk+1     ]*( phi[ijk+1     ]-phi[ijk       ] )
                                     - beta_i[ijk       ]*( phi[ijk       ]-phi[ijk-1     ] )
                                     + beta_j[ijk+STRIDE]*( phi[ijk+STRIDE]-phi[ijk       ] )
                                     - beta_j[ijk       ]*( phi[ijk       ]-phi[ijk-STRIDE] )
                                     + beta_k[ijk+(STRIDE*STRIDE) ]*( phi[ijk+(STRIDE*STRIDE) ]-phi[ijk       ] )
                                     - beta_k[ijk       ]*( phi[ijk       ]-phi[ijk-(STRIDE*STRIDE) ] )
                                      );

                phi_new[ijk] = phi[ijk] - lambda[ijk]*(helmholtz-rhs[ijk]);
            }
        }
    }
}

__global__ void
brick_kernel(unsigned (*grid)[STRIDE/TILEJ][STRIDE/TILEI], Brick3D phi, Brick3D alpha, Brick3D beta_i, Brick3D beta_j,
        Brick3D beta_k, Brick3D phi_new, Brick3D lambda, Brick3D rhs) {
    unsigned k = GZ / TILEK + blockIdx.z;
    unsigned j = GZ / TILEJ + blockIdx.y;
    unsigned i = GZ / TILEI + blockIdx.x;

    unsigned b = grid[k][j][i];

{
  auto *binfo = phi_new.bInfo;
  long neighbor0 = binfo->adj[b][0];
  long neighbor1 = binfo->adj[b][1];
  long neighbor2 = binfo->adj[b][2];
  long neighbor3 = binfo->adj[b][3];
  long neighbor4 = binfo->adj[b][4];
  long neighbor5 = binfo->adj[b][5];
  long neighbor6 = binfo->adj[b][6];
  long neighbor7 = binfo->adj[b][7];
  long neighbor8 = binfo->adj[b][8];
  long neighbor9 = binfo->adj[b][9];
  long neighbor10 = binfo->adj[b][10];
  long neighbor11 = binfo->adj[b][11];
  long neighbor12 = binfo->adj[b][12];
  long neighbor13 = b;
  long neighbor14 = binfo->adj[b][14];
  long neighbor15 = binfo->adj[b][15];
  long neighbor16 = binfo->adj[b][16];
  long neighbor17 = binfo->adj[b][17];
  long neighbor18 = binfo->adj[b][18];
  long neighbor19 = binfo->adj[b][19];
  long neighbor20 = binfo->adj[b][20];
  long neighbor21 = binfo->adj[b][21];
  long neighbor22 = binfo->adj[b][22];
  long neighbor23 = binfo->adj[b][23];
  long neighbor24 = binfo->adj[b][24];
  long neighbor25 = binfo->adj[b][25];
  long neighbor26 = binfo->adj[b][26];
  bElem buf0[16];
  {
    {
      {
        long _cg_rel2 = 0;
        for (long _cg_idx2 = 0; _cg_idx2 < 8; _cg_idx2 += 1)
        {
          long rel = _cg_rel2;
          {
            // New offset [0, 0, 0]
            buf0[0 + rel] = 0;
          }
          {
            // New offset [0, 4, 0]
            buf0[1 + rel] = 0;
          }
          _cg_rel2 += 2;
        }
      }
    }
    {
      bElem _cg_beta_i000_vecbuf;
      bElem _cg_beta_i100_vecbuf;
      bElem _cg_phi000_vecbuf;
      bElem _cg_phi100_vecbuf;
      bElem _cg_phi_100_vecbuf;
      bElem _cg_beta_j010_vecbuf;
      bElem _cg_beta_j000_vecbuf;
      bElem _cg_beta_k001_vecbuf;
      bElem _cg_beta_k000_vecbuf;
      {
        long _cg_rel2 = 0;
        for (long _cg_idx2 = 0; _cg_idx2 < 7; _cg_idx2 += 1)
        {
          long rel = _cg_rel2;
          {
            // New offset [-1, 0, 0]
            bElem _cg_beta_i100_reg;
            bElem _cg_phi100_reg;
            bElem _cg_phi000_reg;
            bElem _cg_beta_j100_reg;
            bElem _cg_beta_k101_reg;
            bElem _cg_beta_k100_reg;
            {
              _cg_beta_i000_vecbuf = beta_i.dat[neighbor13 * beta_i.step + (threadIdx.x + rel * 32)];
              _cg_phi_100_vecbuf = phi.dat[neighbor12 * phi.step + (threadIdx.x + rel * 32)];
              _cg_phi000_vecbuf = phi.dat[neighbor13 * phi.step + (threadIdx.x + rel * 32)];
              _cg_beta_j000_vecbuf = beta_j.dat[neighbor13 * beta_j.step + (threadIdx.x + rel * 32)];
              _cg_beta_k000_vecbuf = beta_k.dat[neighbor13 * beta_k.step + (threadIdx.x + rel * 32)];
              _cg_beta_k001_vecbuf = beta_k.dat[neighbor13 * beta_k.step + 64 + (threadIdx.x + rel * 32)];
              _cg_beta_i100_reg = _cg_beta_i000_vecbuf;
              _cg_phi100_reg = _cg_phi000_vecbuf;
              bElem _cg_vectmp0;
              // merge0 _cg_phi_100_vecbuf ,_cg_phi000_vecbuf, 7 -> _cg_vectmp0
              cu_shl(_cg_vectmp0, _cg_phi_100_vecbuf, _cg_phi000_vecbuf, 1, 8, threadIdx.x & 7);
              _cg_phi000_reg = _cg_vectmp0;
              _cg_beta_j100_reg = _cg_beta_j000_vecbuf;
              _cg_beta_k101_reg = _cg_beta_k001_vecbuf;
              _cg_beta_k100_reg = _cg_beta_k000_vecbuf;
            }
            buf0[0 + rel] -= _cg_beta_i100_reg * _cg_phi100_reg;
            buf0[0 + rel] += _cg_beta_i100_reg * _cg_phi000_reg;
            buf0[0 + rel] -= _cg_beta_j100_reg * _cg_phi100_reg;
            buf0[0 + rel] -= _cg_beta_k101_reg * _cg_phi100_reg;
            buf0[0 + rel] -= _cg_beta_k100_reg * _cg_phi100_reg;
          }
          {
            // New offset [0, 0, 0]
            bElem _cg_beta_i100_reg;
            bElem _cg_phi100_reg;
            bElem _cg_phi000_reg;
            bElem _cg_beta_j010_reg;
            {
              _cg_beta_i100_vecbuf = beta_i.dat[neighbor14 * beta_i.step + (threadIdx.x + rel * 32)];
              _cg_phi100_vecbuf = phi.dat[neighbor14 * phi.step + (threadIdx.x + rel * 32)];
              bElem _cg_vectmp0;
              _cg_vectmp0 = beta_j.dat[neighbor13 * beta_j.step + 32 + (threadIdx.x + rel * 32)];
              bElem _cg_vectmp1;
              _cg_vectmp1 = beta_j.dat[neighbor13 * beta_j.step + (threadIdx.x + rel * 32)];
              // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_beta_j010_vecbuf
              cu_shl(_cg_beta_j010_vecbuf, _cg_vectmp1, _cg_vectmp0, 24, 32, threadIdx.x);
              bElem _cg_vectmp2;
              // merge0 _cg_beta_i000_vecbuf ,_cg_beta_i100_vecbuf, 1 -> _cg_vectmp2
              cu_shl(_cg_vectmp2, _cg_beta_i000_vecbuf, _cg_beta_i100_vecbuf, 7, 8, threadIdx.x & 7);
              _cg_beta_i100_reg = _cg_vectmp2;
              bElem _cg_vectmp3;
              // merge0 _cg_phi000_vecbuf ,_cg_phi100_vecbuf, 1 -> _cg_vectmp3
              cu_shl(_cg_vectmp3, _cg_phi000_vecbuf, _cg_phi100_vecbuf, 7, 8, threadIdx.x & 7);
              _cg_phi100_reg = _cg_vectmp3;
              _cg_phi000_reg = _cg_phi000_vecbuf;
              _cg_beta_j010_reg = _cg_beta_j010_vecbuf;
            }
            buf0[0 + rel] += _cg_beta_i100_reg * _cg_phi100_reg;
            buf0[0 + rel] -= _cg_beta_i100_reg * _cg_phi000_reg;
            buf0[0 + rel] -= _cg_beta_j010_reg * _cg_phi000_reg;
          }
          {
            // New offset [-1, 4, 0]
            bElem _cg_beta_i100_reg;
            bElem _cg_phi100_reg;
            bElem _cg_phi000_reg;
            bElem _cg_beta_j100_reg;
            bElem _cg_beta_k101_reg;
            bElem _cg_beta_k100_reg;
            {
              _cg_beta_i000_vecbuf = beta_i.dat[neighbor13 * beta_i.step + 32 + (threadIdx.x + rel * 32)];
              _cg_phi_100_vecbuf = phi.dat[neighbor12 * phi.step + 32 + (threadIdx.x + rel * 32)];
              _cg_phi000_vecbuf = phi.dat[neighbor13 * phi.step + 32 + (threadIdx.x + rel * 32)];
              _cg_beta_j000_vecbuf = beta_j.dat[neighbor13 * beta_j.step + 32 + (threadIdx.x + rel * 32)];
              _cg_beta_k000_vecbuf = beta_k.dat[neighbor13 * beta_k.step + 32 + (threadIdx.x + rel * 32)];
              _cg_beta_k001_vecbuf = beta_k.dat[neighbor13 * beta_k.step + 96 + (threadIdx.x + rel * 32)];
              _cg_beta_i100_reg = _cg_beta_i000_vecbuf;
              _cg_phi100_reg = _cg_phi000_vecbuf;
              bElem _cg_vectmp0;
              // merge0 _cg_phi_100_vecbuf ,_cg_phi000_vecbuf, 7 -> _cg_vectmp0
              cu_shl(_cg_vectmp0, _cg_phi_100_vecbuf, _cg_phi000_vecbuf, 1, 8, threadIdx.x & 7);
              _cg_phi000_reg = _cg_vectmp0;
              _cg_beta_j100_reg = _cg_beta_j000_vecbuf;
              _cg_beta_k101_reg = _cg_beta_k001_vecbuf;
              _cg_beta_k100_reg = _cg_beta_k000_vecbuf;
            }
            buf0[1 + rel] -= _cg_beta_i100_reg * _cg_phi100_reg;
            buf0[1 + rel] += _cg_beta_i100_reg * _cg_phi000_reg;
            buf0[1 + rel] -= _cg_beta_j100_reg * _cg_phi100_reg;
            buf0[1 + rel] -= _cg_beta_k101_reg * _cg_phi100_reg;
            buf0[1 + rel] -= _cg_beta_k100_reg * _cg_phi100_reg;
          }
          {
            // New offset [0, 4, 0]
            bElem _cg_beta_i100_reg;
            bElem _cg_phi100_reg;
            bElem _cg_phi000_reg;
            bElem _cg_beta_j010_reg;
            {
              _cg_beta_i100_vecbuf = beta_i.dat[neighbor14 * beta_i.step + 32 + (threadIdx.x + rel * 32)];
              _cg_phi100_vecbuf = phi.dat[neighbor14 * phi.step + 32 + (threadIdx.x + rel * 32)];
              bElem _cg_vectmp0;
              _cg_vectmp0 = beta_j.dat[neighbor16 * beta_j.step + (threadIdx.x + rel * 32)];
              bElem _cg_vectmp1;
              _cg_vectmp1 = beta_j.dat[neighbor13 * beta_j.step + 32 + (threadIdx.x + rel * 32)];
              // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_beta_j010_vecbuf
              cu_shl(_cg_beta_j010_vecbuf, _cg_vectmp1, _cg_vectmp0, 24, 32, threadIdx.x);
              bElem _cg_vectmp2;
              // merge0 _cg_beta_i000_vecbuf ,_cg_beta_i100_vecbuf, 1 -> _cg_vectmp2
              cu_shl(_cg_vectmp2, _cg_beta_i000_vecbuf, _cg_beta_i100_vecbuf, 7, 8, threadIdx.x & 7);
              _cg_beta_i100_reg = _cg_vectmp2;
              bElem _cg_vectmp3;
              // merge0 _cg_phi000_vecbuf ,_cg_phi100_vecbuf, 1 -> _cg_vectmp3
              cu_shl(_cg_vectmp3, _cg_phi000_vecbuf, _cg_phi100_vecbuf, 7, 8, threadIdx.x & 7);
              _cg_phi100_reg = _cg_vectmp3;
              _cg_phi000_reg = _cg_phi000_vecbuf;
              _cg_beta_j010_reg = _cg_beta_j010_vecbuf;
            }
            buf0[1 + rel] += _cg_beta_i100_reg * _cg_phi100_reg;
            buf0[1 + rel] -= _cg_beta_i100_reg * _cg_phi000_reg;
            buf0[1 + rel] -= _cg_beta_j010_reg * _cg_phi000_reg;
          }
          _cg_rel2 += 2;
        }
      }
      {
        // New offset [-1, 0, 7]
        bElem _cg_beta_i100_reg;
        bElem _cg_phi100_reg;
        bElem _cg_phi000_reg;
        bElem _cg_beta_j100_reg;
        bElem _cg_beta_k101_reg;
        bElem _cg_beta_k100_reg;
        {
          _cg_beta_i000_vecbuf = beta_i.dat[neighbor13 * beta_i.step + 448 + threadIdx.x];
          _cg_phi_100_vecbuf = phi.dat[neighbor12 * phi.step + 448 + threadIdx.x];
          _cg_phi000_vecbuf = phi.dat[neighbor13 * phi.step + 448 + threadIdx.x];
          _cg_beta_j000_vecbuf = beta_j.dat[neighbor13 * beta_j.step + 448 + threadIdx.x];
          _cg_beta_k000_vecbuf = beta_k.dat[neighbor13 * beta_k.step + 448 + threadIdx.x];
          _cg_beta_k001_vecbuf = beta_k.dat[neighbor22 * beta_k.step + threadIdx.x];
          _cg_beta_i100_reg = _cg_beta_i000_vecbuf;
          _cg_phi100_reg = _cg_phi000_vecbuf;
          bElem _cg_vectmp0;
          // merge0 _cg_phi_100_vecbuf ,_cg_phi000_vecbuf, 7 -> _cg_vectmp0
          cu_shl(_cg_vectmp0, _cg_phi_100_vecbuf, _cg_phi000_vecbuf, 1, 8, threadIdx.x & 7);
          _cg_phi000_reg = _cg_vectmp0;
          _cg_beta_j100_reg = _cg_beta_j000_vecbuf;
          _cg_beta_k101_reg = _cg_beta_k001_vecbuf;
          _cg_beta_k100_reg = _cg_beta_k000_vecbuf;
        }
        buf0[14] -= _cg_beta_i100_reg * _cg_phi100_reg;
        buf0[14] += _cg_beta_i100_reg * _cg_phi000_reg;
        buf0[14] -= _cg_beta_j100_reg * _cg_phi100_reg;
        buf0[14] -= _cg_beta_k101_reg * _cg_phi100_reg;
        buf0[14] -= _cg_beta_k100_reg * _cg_phi100_reg;
      }
      {
        // New offset [0, 0, 7]
        bElem _cg_beta_i100_reg;
        bElem _cg_phi100_reg;
        bElem _cg_phi000_reg;
        bElem _cg_beta_j010_reg;
        {
          _cg_beta_i100_vecbuf = beta_i.dat[neighbor14 * beta_i.step + 448 + threadIdx.x];
          _cg_phi100_vecbuf = phi.dat[neighbor14 * phi.step + 448 + threadIdx.x];
          bElem _cg_vectmp0;
          _cg_vectmp0 = beta_j.dat[neighbor13 * beta_j.step + 480 + threadIdx.x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = beta_j.dat[neighbor13 * beta_j.step + 448 + threadIdx.x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_beta_j010_vecbuf
          cu_shl(_cg_beta_j010_vecbuf, _cg_vectmp1, _cg_vectmp0, 24, 32, threadIdx.x);
          bElem _cg_vectmp2;
          // merge0 _cg_beta_i000_vecbuf ,_cg_beta_i100_vecbuf, 1 -> _cg_vectmp2
          cu_shl(_cg_vectmp2, _cg_beta_i000_vecbuf, _cg_beta_i100_vecbuf, 7, 8, threadIdx.x & 7);
          _cg_beta_i100_reg = _cg_vectmp2;
          bElem _cg_vectmp3;
          // merge0 _cg_phi000_vecbuf ,_cg_phi100_vecbuf, 1 -> _cg_vectmp3
          cu_shl(_cg_vectmp3, _cg_phi000_vecbuf, _cg_phi100_vecbuf, 7, 8, threadIdx.x & 7);
          _cg_phi100_reg = _cg_vectmp3;
          _cg_phi000_reg = _cg_phi000_vecbuf;
          _cg_beta_j010_reg = _cg_beta_j010_vecbuf;
        }
        buf0[14] += _cg_beta_i100_reg * _cg_phi100_reg;
        buf0[14] -= _cg_beta_i100_reg * _cg_phi000_reg;
        buf0[14] -= _cg_beta_j010_reg * _cg_phi000_reg;
      }
      {
        // New offset [-1, 4, 7]
        bElem _cg_beta_i100_reg;
        bElem _cg_phi100_reg;
        bElem _cg_phi000_reg;
        bElem _cg_beta_j100_reg;
        bElem _cg_beta_k101_reg;
        bElem _cg_beta_k100_reg;
        {
          _cg_beta_i000_vecbuf = beta_i.dat[neighbor13 * beta_i.step + 480 + threadIdx.x];
          _cg_phi_100_vecbuf = phi.dat[neighbor12 * phi.step + 480 + threadIdx.x];
          _cg_phi000_vecbuf = phi.dat[neighbor13 * phi.step + 480 + threadIdx.x];
          _cg_beta_j000_vecbuf = beta_j.dat[neighbor13 * beta_j.step + 480 + threadIdx.x];
          _cg_beta_k000_vecbuf = beta_k.dat[neighbor13 * beta_k.step + 480 + threadIdx.x];
          _cg_beta_k001_vecbuf = beta_k.dat[neighbor22 * beta_k.step + 32 + threadIdx.x];
          _cg_beta_i100_reg = _cg_beta_i000_vecbuf;
          _cg_phi100_reg = _cg_phi000_vecbuf;
          bElem _cg_vectmp0;
          // merge0 _cg_phi_100_vecbuf ,_cg_phi000_vecbuf, 7 -> _cg_vectmp0
          cu_shl(_cg_vectmp0, _cg_phi_100_vecbuf, _cg_phi000_vecbuf, 1, 8, threadIdx.x & 7);
          _cg_phi000_reg = _cg_vectmp0;
          _cg_beta_j100_reg = _cg_beta_j000_vecbuf;
          _cg_beta_k101_reg = _cg_beta_k001_vecbuf;
          _cg_beta_k100_reg = _cg_beta_k000_vecbuf;
        }
        buf0[15] -= _cg_beta_i100_reg * _cg_phi100_reg;
        buf0[15] += _cg_beta_i100_reg * _cg_phi000_reg;
        buf0[15] -= _cg_beta_j100_reg * _cg_phi100_reg;
        buf0[15] -= _cg_beta_k101_reg * _cg_phi100_reg;
        buf0[15] -= _cg_beta_k100_reg * _cg_phi100_reg;
      }
      {
        // New offset [0, 4, 7]
        bElem _cg_beta_i100_reg;
        bElem _cg_phi100_reg;
        bElem _cg_phi000_reg;
        bElem _cg_beta_j010_reg;
        {
          _cg_beta_i100_vecbuf = beta_i.dat[neighbor14 * beta_i.step + 480 + threadIdx.x];
          _cg_phi100_vecbuf = phi.dat[neighbor14 * phi.step + 480 + threadIdx.x];
          bElem _cg_vectmp0;
          _cg_vectmp0 = beta_j.dat[neighbor16 * beta_j.step + 448 + threadIdx.x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = beta_j.dat[neighbor13 * beta_j.step + 480 + threadIdx.x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_beta_j010_vecbuf
          cu_shl(_cg_beta_j010_vecbuf, _cg_vectmp1, _cg_vectmp0, 24, 32, threadIdx.x);
          bElem _cg_vectmp2;
          // merge0 _cg_beta_i000_vecbuf ,_cg_beta_i100_vecbuf, 1 -> _cg_vectmp2
          cu_shl(_cg_vectmp2, _cg_beta_i000_vecbuf, _cg_beta_i100_vecbuf, 7, 8, threadIdx.x & 7);
          _cg_beta_i100_reg = _cg_vectmp2;
          bElem _cg_vectmp3;
          // merge0 _cg_phi000_vecbuf ,_cg_phi100_vecbuf, 1 -> _cg_vectmp3
          cu_shl(_cg_vectmp3, _cg_phi000_vecbuf, _cg_phi100_vecbuf, 7, 8, threadIdx.x & 7);
          _cg_phi100_reg = _cg_vectmp3;
          _cg_phi000_reg = _cg_phi000_vecbuf;
          _cg_beta_j010_reg = _cg_beta_j010_vecbuf;
        }
        buf0[15] += _cg_beta_i100_reg * _cg_phi100_reg;
        buf0[15] -= _cg_beta_i100_reg * _cg_phi000_reg;
        buf0[15] -= _cg_beta_j010_reg * _cg_phi000_reg;
      }
    }
    {
      bElem _cg_beta_j000_vecbuf;
      bElem _cg_beta_j010_vecbuf;
      bElem _cg_phi000_vecbuf;
      bElem _cg_beta_k000_vecbuf;
      bElem _cg_beta_k001_vecbuf;
      {
        // New offset [-1, 0, -1]
        bElem _cg_beta_k101_reg;
        bElem _cg_phi100_reg;
        {
          _cg_beta_k001_vecbuf = beta_k.dat[neighbor13 * beta_k.step + threadIdx.x];
          _cg_phi000_vecbuf = phi.dat[neighbor4 * phi.step + 448 + threadIdx.x];
          _cg_beta_k101_reg = _cg_beta_k001_vecbuf;
          _cg_phi100_reg = _cg_phi000_vecbuf;
        }
        buf0[0] += _cg_beta_k101_reg * _cg_phi100_reg;
      }
      {
        // New offset [-1, 4, -1]
        bElem _cg_beta_k101_reg;
        bElem _cg_phi100_reg;
        {
          _cg_beta_k001_vecbuf = beta_k.dat[neighbor13 * beta_k.step + 32 + threadIdx.x];
          _cg_phi000_vecbuf = phi.dat[neighbor4 * phi.step + 480 + threadIdx.x];
          _cg_beta_k101_reg = _cg_beta_k001_vecbuf;
          _cg_phi100_reg = _cg_phi000_vecbuf;
        }
        buf0[1] += _cg_beta_k101_reg * _cg_phi100_reg;
      }
      {
        // New offset [0, -1, 0]
        bElem _cg_beta_j010_reg;
        bElem _cg_phi000_reg;
        {
          _cg_beta_j010_vecbuf = beta_j.dat[neighbor13 * beta_j.step + threadIdx.x];
          bElem _cg_vectmp0;
          _cg_vectmp0 = phi.dat[neighbor13 * phi.step + threadIdx.x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = phi.dat[neighbor10 * phi.step + 32 + threadIdx.x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 3 -> _cg_phi000_vecbuf
          cu_shl(_cg_phi000_vecbuf, _cg_vectmp1, _cg_vectmp0, 8, 32, threadIdx.x);
          _cg_beta_j010_reg = _cg_beta_j010_vecbuf;
          _cg_phi000_reg = _cg_phi000_vecbuf;
        }
        buf0[0] += _cg_beta_j010_reg * _cg_phi000_reg;
      }
      {
        // New offset [-1, 0, 0]
        bElem _cg_beta_k101_reg;
        bElem _cg_phi100_reg;
        {
          _cg_beta_k001_vecbuf = beta_k.dat[neighbor13 * beta_k.step + 64 + threadIdx.x];
          _cg_phi000_vecbuf = phi.dat[neighbor13 * phi.step + threadIdx.x];
          _cg_beta_k101_reg = _cg_beta_k001_vecbuf;
          _cg_phi100_reg = _cg_phi000_vecbuf;
        }
        buf0[2] += _cg_beta_k101_reg * _cg_phi100_reg;
      }
      {
        // New offset [-1, 1, 0]
        bElem _cg_beta_j100_reg;
        bElem _cg_phi100_reg;
        {
          bElem _cg_vectmp0;
          _cg_vectmp0 = beta_j.dat[neighbor13 * beta_j.step + 32 + threadIdx.x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = beta_j.dat[neighbor13 * beta_j.step + threadIdx.x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_beta_j000_vecbuf
          cu_shl(_cg_beta_j000_vecbuf, _cg_vectmp1, _cg_vectmp0, 24, 32, threadIdx.x);
          bElem _cg_vectmp2;
          _cg_vectmp2 = phi.dat[neighbor13 * phi.step + 32 + threadIdx.x];
          bElem _cg_vectmp3;
          _cg_vectmp3 = phi.dat[neighbor13 * phi.step + threadIdx.x];
          // merge1 _cg_vectmp3 ,_cg_vectmp2, 1 -> _cg_phi000_vecbuf
          cu_shl(_cg_phi000_vecbuf, _cg_vectmp3, _cg_vectmp2, 24, 32, threadIdx.x);
          _cg_beta_j100_reg = _cg_beta_j000_vecbuf;
          _cg_phi100_reg = _cg_phi000_vecbuf;
        }
        buf0[0] += _cg_beta_j100_reg * _cg_phi100_reg;
      }
      {
        // New offset [0, 3, 0]
        bElem _cg_beta_j010_reg;
        bElem _cg_phi000_reg;
        {
          _cg_beta_j010_vecbuf = beta_j.dat[neighbor13 * beta_j.step + 32 + threadIdx.x];
          bElem _cg_vectmp0;
          _cg_vectmp0 = phi.dat[neighbor13 * phi.step + 32 + threadIdx.x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = phi.dat[neighbor13 * phi.step + threadIdx.x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 3 -> _cg_phi000_vecbuf
          cu_shl(_cg_phi000_vecbuf, _cg_vectmp1, _cg_vectmp0, 8, 32, threadIdx.x);
          _cg_beta_j010_reg = _cg_beta_j010_vecbuf;
          _cg_phi000_reg = _cg_phi000_vecbuf;
        }
        buf0[1] += _cg_beta_j010_reg * _cg_phi000_reg;
      }
      {
        // New offset [-1, 4, 0]
        bElem _cg_beta_k101_reg;
        bElem _cg_phi100_reg;
        {
          _cg_beta_k001_vecbuf = beta_k.dat[neighbor13 * beta_k.step + 96 + threadIdx.x];
          _cg_phi000_vecbuf = phi.dat[neighbor13 * phi.step + 32 + threadIdx.x];
          _cg_beta_k101_reg = _cg_beta_k001_vecbuf;
          _cg_phi100_reg = _cg_phi000_vecbuf;
        }
        buf0[3] += _cg_beta_k101_reg * _cg_phi100_reg;
      }
      {
        // New offset [-1, 5, 0]
        bElem _cg_beta_j100_reg;
        bElem _cg_phi100_reg;
        {
          bElem _cg_vectmp0;
          _cg_vectmp0 = beta_j.dat[neighbor16 * beta_j.step + threadIdx.x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = beta_j.dat[neighbor13 * beta_j.step + 32 + threadIdx.x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_beta_j000_vecbuf
          cu_shl(_cg_beta_j000_vecbuf, _cg_vectmp1, _cg_vectmp0, 24, 32, threadIdx.x);
          bElem _cg_vectmp2;
          _cg_vectmp2 = phi.dat[neighbor16 * phi.step + threadIdx.x];
          bElem _cg_vectmp3;
          _cg_vectmp3 = phi.dat[neighbor13 * phi.step + 32 + threadIdx.x];
          // merge1 _cg_vectmp3 ,_cg_vectmp2, 1 -> _cg_phi000_vecbuf
          cu_shl(_cg_phi000_vecbuf, _cg_vectmp3, _cg_vectmp2, 24, 32, threadIdx.x);
          _cg_beta_j100_reg = _cg_beta_j000_vecbuf;
          _cg_phi100_reg = _cg_phi000_vecbuf;
        }
        buf0[1] += _cg_beta_j100_reg * _cg_phi100_reg;
      }
      {
        long _cg_rel2 = 0;
        for (long _cg_idx2 = 0; _cg_idx2 < 6; _cg_idx2 += 1)
        {
          long rel = _cg_rel2;
          {
            // New offset [0, -1, 1]
            bElem _cg_beta_j010_reg;
            bElem _cg_phi000_reg;
            {
              _cg_beta_j010_vecbuf = beta_j.dat[neighbor13 * beta_j.step + 64 + (threadIdx.x + rel * 32)];
              bElem _cg_vectmp0;
              _cg_vectmp0 = phi.dat[neighbor13 * phi.step + 64 + (threadIdx.x + rel * 32)];
              bElem _cg_vectmp1;
              _cg_vectmp1 = phi.dat[neighbor10 * phi.step + 96 + (threadIdx.x + rel * 32)];
              // merge1 _cg_vectmp1 ,_cg_vectmp0, 3 -> _cg_phi000_vecbuf
              cu_shl(_cg_phi000_vecbuf, _cg_vectmp1, _cg_vectmp0, 8, 32, threadIdx.x);
              _cg_beta_j010_reg = _cg_beta_j010_vecbuf;
              _cg_phi000_reg = _cg_phi000_vecbuf;
            }
            buf0[2 + rel] += _cg_beta_j010_reg * _cg_phi000_reg;
          }
          {
            // New offset [-1, 0, 1]
            bElem _cg_beta_k100_reg;
            bElem _cg_phi100_reg;
            bElem _cg_beta_k101_reg;
            {
              _cg_beta_k000_vecbuf = beta_k.dat[neighbor13 * beta_k.step + 64 + (threadIdx.x + rel * 32)];
              _cg_beta_k001_vecbuf = beta_k.dat[neighbor13 * beta_k.step + 128 + (threadIdx.x + rel * 32)];
              _cg_phi000_vecbuf = phi.dat[neighbor13 * phi.step + 64 + (threadIdx.x + rel * 32)];
              _cg_beta_k100_reg = _cg_beta_k000_vecbuf;
              _cg_beta_k101_reg = _cg_beta_k001_vecbuf;
              _cg_phi100_reg = _cg_phi000_vecbuf;
            }
            buf0[0 + rel] += _cg_beta_k100_reg * _cg_phi100_reg;
            buf0[4 + rel] += _cg_beta_k101_reg * _cg_phi100_reg;
          }
          {
            // New offset [-1, 1, 1]
            bElem _cg_beta_j100_reg;
            bElem _cg_phi100_reg;
            {
              bElem _cg_vectmp0;
              _cg_vectmp0 = beta_j.dat[neighbor13 * beta_j.step + 96 + (threadIdx.x + rel * 32)];
              bElem _cg_vectmp1;
              _cg_vectmp1 = beta_j.dat[neighbor13 * beta_j.step + 64 + (threadIdx.x + rel * 32)];
              // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_beta_j000_vecbuf
              cu_shl(_cg_beta_j000_vecbuf, _cg_vectmp1, _cg_vectmp0, 24, 32, threadIdx.x);
              bElem _cg_vectmp2;
              _cg_vectmp2 = phi.dat[neighbor13 * phi.step + 96 + (threadIdx.x + rel * 32)];
              bElem _cg_vectmp3;
              _cg_vectmp3 = phi.dat[neighbor13 * phi.step + 64 + (threadIdx.x + rel * 32)];
              // merge1 _cg_vectmp3 ,_cg_vectmp2, 1 -> _cg_phi000_vecbuf
              cu_shl(_cg_phi000_vecbuf, _cg_vectmp3, _cg_vectmp2, 24, 32, threadIdx.x);
              _cg_beta_j100_reg = _cg_beta_j000_vecbuf;
              _cg_phi100_reg = _cg_phi000_vecbuf;
            }
            buf0[2 + rel] += _cg_beta_j100_reg * _cg_phi100_reg;
          }
          {
            // New offset [0, 3, 1]
            bElem _cg_beta_j010_reg;
            bElem _cg_phi000_reg;
            {
              _cg_beta_j010_vecbuf = beta_j.dat[neighbor13 * beta_j.step + 96 + (threadIdx.x + rel * 32)];
              bElem _cg_vectmp0;
              _cg_vectmp0 = phi.dat[neighbor13 * phi.step + 96 + (threadIdx.x + rel * 32)];
              bElem _cg_vectmp1;
              _cg_vectmp1 = phi.dat[neighbor13 * phi.step + 64 + (threadIdx.x + rel * 32)];
              // merge1 _cg_vectmp1 ,_cg_vectmp0, 3 -> _cg_phi000_vecbuf
              cu_shl(_cg_phi000_vecbuf, _cg_vectmp1, _cg_vectmp0, 8, 32, threadIdx.x);
              _cg_beta_j010_reg = _cg_beta_j010_vecbuf;
              _cg_phi000_reg = _cg_phi000_vecbuf;
            }
            buf0[3 + rel] += _cg_beta_j010_reg * _cg_phi000_reg;
          }
          {
            // New offset [-1, 4, 1]
            bElem _cg_beta_k100_reg;
            bElem _cg_phi100_reg;
            bElem _cg_beta_k101_reg;
            {
              _cg_beta_k000_vecbuf = beta_k.dat[neighbor13 * beta_k.step + 96 + (threadIdx.x + rel * 32)];
              _cg_beta_k001_vecbuf = beta_k.dat[neighbor13 * beta_k.step + 160 + (threadIdx.x + rel * 32)];
              _cg_phi000_vecbuf = phi.dat[neighbor13 * phi.step + 96 + (threadIdx.x + rel * 32)];
              _cg_beta_k100_reg = _cg_beta_k000_vecbuf;
              _cg_beta_k101_reg = _cg_beta_k001_vecbuf;
              _cg_phi100_reg = _cg_phi000_vecbuf;
            }
            buf0[1 + rel] += _cg_beta_k100_reg * _cg_phi100_reg;
            buf0[5 + rel] += _cg_beta_k101_reg * _cg_phi100_reg;
          }
          {
            // New offset [-1, 5, 1]
            bElem _cg_beta_j100_reg;
            bElem _cg_phi100_reg;
            {
              bElem _cg_vectmp0;
              _cg_vectmp0 = beta_j.dat[neighbor16 * beta_j.step + 64 + (threadIdx.x + rel * 32)];
              bElem _cg_vectmp1;
              _cg_vectmp1 = beta_j.dat[neighbor13 * beta_j.step + 96 + (threadIdx.x + rel * 32)];
              // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_beta_j000_vecbuf
              cu_shl(_cg_beta_j000_vecbuf, _cg_vectmp1, _cg_vectmp0, 24, 32, threadIdx.x);
              bElem _cg_vectmp2;
              _cg_vectmp2 = phi.dat[neighbor16 * phi.step + 64 + (threadIdx.x + rel * 32)];
              bElem _cg_vectmp3;
              _cg_vectmp3 = phi.dat[neighbor13 * phi.step + 96 + (threadIdx.x + rel * 32)];
              // merge1 _cg_vectmp3 ,_cg_vectmp2, 1 -> _cg_phi000_vecbuf
              cu_shl(_cg_phi000_vecbuf, _cg_vectmp3, _cg_vectmp2, 24, 32, threadIdx.x);
              _cg_beta_j100_reg = _cg_beta_j000_vecbuf;
              _cg_phi100_reg = _cg_phi000_vecbuf;
            }
            buf0[3 + rel] += _cg_beta_j100_reg * _cg_phi100_reg;
          }
          _cg_rel2 += 2;
        }
      }
      {
        // New offset [0, -1, 7]
        bElem _cg_beta_j010_reg;
        bElem _cg_phi000_reg;
        {
          _cg_beta_j010_vecbuf = beta_j.dat[neighbor13 * beta_j.step + 448 + threadIdx.x];
          bElem _cg_vectmp0;
          _cg_vectmp0 = phi.dat[neighbor13 * phi.step + 448 + threadIdx.x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = phi.dat[neighbor10 * phi.step + 480 + threadIdx.x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 3 -> _cg_phi000_vecbuf
          cu_shl(_cg_phi000_vecbuf, _cg_vectmp1, _cg_vectmp0, 8, 32, threadIdx.x);
          _cg_beta_j010_reg = _cg_beta_j010_vecbuf;
          _cg_phi000_reg = _cg_phi000_vecbuf;
        }
        buf0[14] += _cg_beta_j010_reg * _cg_phi000_reg;
      }
      {
        // New offset [-1, 0, 7]
        bElem _cg_beta_k100_reg;
        bElem _cg_phi100_reg;
        {
          _cg_beta_k000_vecbuf = beta_k.dat[neighbor13 * beta_k.step + 448 + threadIdx.x];
          _cg_phi000_vecbuf = phi.dat[neighbor13 * phi.step + 448 + threadIdx.x];
          _cg_beta_k100_reg = _cg_beta_k000_vecbuf;
          _cg_phi100_reg = _cg_phi000_vecbuf;
        }
        buf0[12] += _cg_beta_k100_reg * _cg_phi100_reg;
      }
      {
        // New offset [-1, 1, 7]
        bElem _cg_beta_j100_reg;
        bElem _cg_phi100_reg;
        {
          bElem _cg_vectmp0;
          _cg_vectmp0 = beta_j.dat[neighbor13 * beta_j.step + 480 + threadIdx.x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = beta_j.dat[neighbor13 * beta_j.step + 448 + threadIdx.x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_beta_j000_vecbuf
          cu_shl(_cg_beta_j000_vecbuf, _cg_vectmp1, _cg_vectmp0, 24, 32, threadIdx.x);
          bElem _cg_vectmp2;
          _cg_vectmp2 = phi.dat[neighbor13 * phi.step + 480 + threadIdx.x];
          bElem _cg_vectmp3;
          _cg_vectmp3 = phi.dat[neighbor13 * phi.step + 448 + threadIdx.x];
          // merge1 _cg_vectmp3 ,_cg_vectmp2, 1 -> _cg_phi000_vecbuf
          cu_shl(_cg_phi000_vecbuf, _cg_vectmp3, _cg_vectmp2, 24, 32, threadIdx.x);
          _cg_beta_j100_reg = _cg_beta_j000_vecbuf;
          _cg_phi100_reg = _cg_phi000_vecbuf;
        }
        buf0[14] += _cg_beta_j100_reg * _cg_phi100_reg;
      }
      {
        // New offset [0, 3, 7]
        bElem _cg_beta_j010_reg;
        bElem _cg_phi000_reg;
        {
          _cg_beta_j010_vecbuf = beta_j.dat[neighbor13 * beta_j.step + 480 + threadIdx.x];
          bElem _cg_vectmp0;
          _cg_vectmp0 = phi.dat[neighbor13 * phi.step + 480 + threadIdx.x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = phi.dat[neighbor13 * phi.step + 448 + threadIdx.x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 3 -> _cg_phi000_vecbuf
          cu_shl(_cg_phi000_vecbuf, _cg_vectmp1, _cg_vectmp0, 8, 32, threadIdx.x);
          _cg_beta_j010_reg = _cg_beta_j010_vecbuf;
          _cg_phi000_reg = _cg_phi000_vecbuf;
        }
        buf0[15] += _cg_beta_j010_reg * _cg_phi000_reg;
      }
      {
        // New offset [-1, 4, 7]
        bElem _cg_beta_k100_reg;
        bElem _cg_phi100_reg;
        {
          _cg_beta_k000_vecbuf = _cg_beta_k001_vecbuf;
          _cg_phi000_vecbuf = phi.dat[neighbor13 * phi.step + 480 + threadIdx.x];
          _cg_beta_k100_reg = _cg_beta_k000_vecbuf;
          _cg_phi100_reg = _cg_phi000_vecbuf;
        }
        buf0[13] += _cg_beta_k100_reg * _cg_phi100_reg;
      }
      {
        // New offset [-1, 5, 7]
        bElem _cg_beta_j100_reg;
        bElem _cg_phi100_reg;
        {
          bElem _cg_vectmp0;
          _cg_vectmp0 = beta_j.dat[neighbor16 * beta_j.step + 448 + threadIdx.x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = beta_j.dat[neighbor13 * beta_j.step + 480 + threadIdx.x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_beta_j000_vecbuf
          cu_shl(_cg_beta_j000_vecbuf, _cg_vectmp1, _cg_vectmp0, 24, 32, threadIdx.x);
          bElem _cg_vectmp2;
          _cg_vectmp2 = phi.dat[neighbor16 * phi.step + 448 + threadIdx.x];
          bElem _cg_vectmp3;
          _cg_vectmp3 = phi.dat[neighbor13 * phi.step + 480 + threadIdx.x];
          // merge1 _cg_vectmp3 ,_cg_vectmp2, 1 -> _cg_phi000_vecbuf
          cu_shl(_cg_phi000_vecbuf, _cg_vectmp3, _cg_vectmp2, 24, 32, threadIdx.x);
          _cg_beta_j100_reg = _cg_beta_j000_vecbuf;
          _cg_phi100_reg = _cg_phi000_vecbuf;
        }
        buf0[15] += _cg_beta_j100_reg * _cg_phi100_reg;
      }
      {
        // New offset [-1, 0, 8]
        bElem _cg_beta_k100_reg;
        bElem _cg_phi100_reg;
        {
          _cg_beta_k000_vecbuf = beta_k.dat[neighbor22 * beta_k.step + threadIdx.x];
          _cg_phi000_vecbuf = phi.dat[neighbor22 * phi.step + threadIdx.x];
          _cg_beta_k100_reg = _cg_beta_k000_vecbuf;
          _cg_phi100_reg = _cg_phi000_vecbuf;
        }
        buf0[14] += _cg_beta_k100_reg * _cg_phi100_reg;
      }
      {
        // New offset [-1, 4, 8]
        bElem _cg_beta_k100_reg;
        bElem _cg_phi100_reg;
        {
          _cg_beta_k000_vecbuf = beta_k.dat[neighbor22 * beta_k.step + 32 + threadIdx.x];
          _cg_phi000_vecbuf = phi.dat[neighbor22 * phi.step + 32 + threadIdx.x];
          _cg_beta_k100_reg = _cg_beta_k000_vecbuf;
          _cg_phi100_reg = _cg_phi000_vecbuf;
        }
        buf0[15] += _cg_beta_k100_reg * _cg_phi100_reg;
      }
    }
  }
  bElem buf1[16];
  {
    {
      bElem _cg_phi000_vecbuf;
      bElem _cg_lambda000_vecbuf;
      bElem _cg_alpha000_vecbuf;
      bElem _cg_rhs000_vecbuf;
      {
        long _cg_rel2 = 0;
        for (long _cg_idx2 = 0; _cg_idx2 < 8; _cg_idx2 += 1)
        {
          long rel = _cg_rel2;
          {
            // New offset [0, 0, 0]
            bElem _cg_phi000_reg;
            bElem _cg_lambda000_reg;
            bElem _cg_alpha000_reg;
            bElem _cg_rhs000_reg;
            {
              _cg_phi000_vecbuf = phi.dat[neighbor13 * phi.step + (threadIdx.x + rel * 32)];
              _cg_lambda000_vecbuf = lambda.dat[neighbor13 * lambda.step + (threadIdx.x + rel * 32)];
              _cg_alpha000_vecbuf = alpha.dat[neighbor13 * alpha.step + (threadIdx.x + rel * 32)];
              _cg_rhs000_vecbuf = rhs.dat[neighbor13 * rhs.step + (threadIdx.x + rel * 32)];
              _cg_phi000_reg = _cg_phi000_vecbuf;
              _cg_lambda000_reg = _cg_lambda000_vecbuf;
              _cg_alpha000_reg = _cg_alpha000_vecbuf;
              _cg_rhs000_reg = _cg_rhs000_vecbuf;
            }
            buf1[0 + rel] = _cg_phi000_reg - _cg_lambda000_reg * (_cg_alpha000_reg * _cg_phi000_reg - H2INV * buf0[0 + rel] - _cg_rhs000_reg);
          }
          {
            // New offset [0, 4, 0]
            bElem _cg_phi000_reg;
            bElem _cg_lambda000_reg;
            bElem _cg_alpha000_reg;
            bElem _cg_rhs000_reg;
            {
              _cg_phi000_vecbuf = phi.dat[neighbor13 * phi.step + 32 + (threadIdx.x + rel * 32)];
              _cg_lambda000_vecbuf = lambda.dat[neighbor13 * lambda.step + 32 + (threadIdx.x + rel * 32)];
              _cg_alpha000_vecbuf = alpha.dat[neighbor13 * alpha.step + 32 + (threadIdx.x + rel * 32)];
              _cg_rhs000_vecbuf = rhs.dat[neighbor13 * rhs.step + 32 + (threadIdx.x + rel * 32)];
              _cg_phi000_reg = _cg_phi000_vecbuf;
              _cg_lambda000_reg = _cg_lambda000_vecbuf;
              _cg_alpha000_reg = _cg_alpha000_vecbuf;
              _cg_rhs000_reg = _cg_rhs000_vecbuf;
            }
            buf1[1 + rel] = _cg_phi000_reg - _cg_lambda000_reg * (_cg_alpha000_reg * _cg_phi000_reg - H2INV * buf0[1 + rel] - _cg_rhs000_reg);
          }
          _cg_rel2 += 2;
        }
      }
    }
    bElem *phi_new_ref = &phi_new.dat[neighbor13 * phi_new.step];
    for (long sti = 0; sti < 16; ++sti)
    {
      phi_new_ref[sti * 32 + threadIdx.x] = buf1[sti];
    }
  }
}

}

int main() {
    // allocations
    bElem *c = randomArray({3});
    bElem *c_dev;
    copyToDevice({3}, c_dev, c);

    auto phi_arr = randomArray({STRIDE, STRIDE, STRIDE});
    bElem *phi_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, phi_dev, phi_arr);

    auto phi_new_arr = zeroArray({STRIDE, STRIDE, STRIDE});
    bElem *phi_new_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, phi_new_dev, phi_new_arr);

    auto alpha_arr = randomArray({STRIDE, STRIDE, STRIDE});
    bElem *alpha_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, alpha_dev, alpha_arr);

    auto beta_i_arr = randomArray({STRIDE, STRIDE, STRIDE});
    bElem *beta_i_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, beta_i_dev, beta_i_arr);

    auto beta_j_arr = randomArray({STRIDE, STRIDE, STRIDE});
    bElem *beta_j_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, beta_j_dev, beta_j_arr);

    auto beta_k_arr = randomArray({STRIDE, STRIDE, STRIDE});
    bElem *beta_k_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, beta_k_dev, beta_k_arr);

    auto rhs_arr = randomArray({STRIDE, STRIDE, STRIDE});
    bElem *rhs_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, rhs_dev, rhs_arr);

    auto lambda_arr = randomArray({STRIDE, STRIDE, STRIDE});
    bElem *lambda_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, lambda_dev, lambda_arr);

    // CPU
    {

        for(int k=GZ;k<STRIDE-GZ;k++){
            for(int j=GZ;j<STRIDE-GZ;j++){
                for(int i=GZ;i<STRIDE-GZ;i++){
                    int ijk = i + j*STRIDE + k*STRIDE*STRIDE;

                    { // color signifies red or black case
                      double helmholtz = alpha_arr[ijk]*phi_arr[ijk]
                                       - H2INV*(
                                           beta_i_arr[ijk+1     ]*( phi_arr[ijk+1     ]-phi_arr[ijk       ] )
                                         - beta_i_arr[ijk       ]*( phi_arr[ijk       ]-phi_arr[ijk-1     ] )
                                         + beta_j_arr[ijk+STRIDE]*( phi_arr[ijk+STRIDE]-phi_arr[ijk       ] )
                                         - beta_j_arr[ijk       ]*( phi_arr[ijk       ]-phi_arr[ijk-STRIDE] )
                                         + beta_k_arr[ijk+(STRIDE*STRIDE) ]*( phi_arr[ijk+(STRIDE*STRIDE) ]-phi_arr[ijk       ] )
                                         - beta_k_arr[ijk       ]*( phi_arr[ijk       ]-phi_arr[ijk-(STRIDE*STRIDE) ] )
                                            );

                      phi_new_arr[ijk] = phi_arr[ijk] - lambda_arr[ijk]*(helmholtz-rhs_arr[ijk]);
                    }
                }
            }
        }
    }

    // CUDA
    {
        auto compute = [&]() -> void {
            long numOfThreads = STRIDE;
            // long numOfBlocks = ceil(STRIDE/numOfThreads);
            long numOfBlocks = ceil(STRIDE*STRIDE*STRIDE/(numOfThreads*STRIDE*STRIDE)); // Unroll on i

            dim3 dimBlock(numOfThreads);
            dim3 dimGrid(numOfBlocks);
            
            GSRBKernel<<<dimGrid, dimBlock>>> (phi_dev, phi_new_dev, rhs_dev, alpha_dev, beta_i_dev, 
                                         beta_j_dev , beta_k_dev , lambda_dev, 0);
        };
        
        compute();
        auto phi_new_cuda = zeroArray({STRIDE, STRIDE, STRIDE});
        copyFromDevice({STRIDE, STRIDE, STRIDE}, phi_new_cuda, phi_new_dev);

        if (!compareArray({STRIDE, STRIDE, STRIDE}, (bElem *)phi_new_arr, (bElem *)phi_new_cuda))
            std::cout << "arr don't match" << std::endl;
    }

    hipDeviceSynchronize();

    hipFree(phi_dev);
    hipFree(phi_new_dev);
    hipFree(alpha_dev);
    hipFree(beta_i_dev);
    hipFree(beta_j_dev);
    hipFree(beta_k_dev);
    hipFree(rhs_dev);
    hipFree(lambda_dev);
    
    {
        unsigned *grid_ptr;
        unsigned bSize = TILEK * TILEJ * TILEI;

        auto bInfo = init_grid<3>(grid_ptr, {STRIDE/TILEK, STRIDE/TILEJ, STRIDE/TILEI});

        unsigned *grid_dev;
        copyToDevice({STRIDE/TILEK, STRIDE/TILEJ, STRIDE/TILEI}, grid_dev, grid_ptr);

        auto bStorage = BrickStorage::allocate(bInfo.nbricks, bSize * 8);

        Brick3D phi_bri(&bInfo, &bStorage, 0);
        Brick3D alpha_bri(&bInfo, &bStorage, bSize);
        Brick3D beta_i_bri(&bInfo, &bStorage, bSize * 2);
        Brick3D beta_j_bri(&bInfo, &bStorage, bSize * 3);
        Brick3D beta_k_bri(&bInfo, &bStorage, bSize * 4);
        Brick3D phi_new_bri(&bInfo, &bStorage, bSize * 5);
        Brick3D lambda_bri(&bInfo, &bStorage, bSize * 6);
        Brick3D rhs_bri(&bInfo, &bStorage, bSize * 7);

        BrickInfo<3> *bInfo_dev;
        auto _bInfo_dev = movBrickInfo(bInfo, hipMemcpyHostToDevice);
        {
            unsigned size = sizeof(BrickInfo<3>);
            hipMalloc(&bInfo_dev, size);
            hipMemcpy(bInfo_dev, &_bInfo_dev, size, hipMemcpyHostToDevice);
        }

        copyBrick<3>({STRIDE, STRIDE, STRIDE}, phi_arr, grid_ptr, phi_bri);
        copyBrick<3>({STRIDE, STRIDE, STRIDE}, alpha_arr, grid_ptr, alpha_bri);
        copyBrick<3>({STRIDE, STRIDE, STRIDE}, beta_i_arr, grid_ptr, beta_i_bri);
        copyBrick<3>({STRIDE, STRIDE, STRIDE}, beta_j_arr, grid_ptr, beta_j_bri);
        copyBrick<3>({STRIDE, STRIDE, STRIDE}, beta_k_arr, grid_ptr, beta_k_bri);
        copyBrick<3>({STRIDE, STRIDE, STRIDE}, phi_new_arr, grid_ptr, phi_new_bri);
        copyBrick<3>({STRIDE, STRIDE, STRIDE}, lambda_arr, grid_ptr, lambda_bri);
        copyBrick<3>({STRIDE, STRIDE, STRIDE}, rhs_arr, grid_ptr, rhs_bri);

        BrickStorage *bStorage_dev;
        BrickStorage _bStorage_dev = movBrickStorage(bStorage, hipMemcpyHostToDevice);
        {
            unsigned size = sizeof(BrickStorage);
            hipMalloc(&bStorage_dev, size);
            hipMemcpy(bStorage_dev, &_bStorage_dev, size, hipMemcpyHostToDevice);
        }

        auto compute = [&]() -> void {
            Brick3D phi(bInfo_dev, &_bStorage_dev, 0);
            Brick3D alpha(bInfo_dev, &_bStorage_dev, bSize);
            Brick3D beta_i(bInfo_dev, &_bStorage_dev, bSize * 2);
            Brick3D beta_j(bInfo_dev, &_bStorage_dev, bSize * 3);
            Brick3D beta_k(bInfo_dev, &_bStorage_dev, bSize * 4);
            Brick3D phi_new(bInfo_dev, &_bStorage_dev, bSize * 5);
            Brick3D lambda(bInfo_dev, &bStorage, bSize * 6);
            Brick3D rhs(bInfo_dev, &bStorage, bSize * 7);
           
            phi.bStorage = bStorage_dev;
            alpha.bStorage = bStorage_dev;
            beta_i.bStorage = bStorage_dev;
            beta_j.bStorage = bStorage_dev;
            beta_k.bStorage = bStorage_dev;
            phi_new.bStorage = bStorage_dev;
            lambda.bStorage = bStorage_dev;
            rhs.bStorage = bStorage_dev;

            auto grid = (unsigned (*)[STRIDE/TILEJ][STRIDE/TILEI]) grid_dev;
            dim3 block(N/TILEI, N/TILEJ, N/TILEK), thread(32);
            
            brick_kernel<<< block, thread >>>(grid, phi, alpha, beta_i, beta_j, beta_k, phi_new, lambda, rhs);
        };

        compute();

        hipDeviceSynchronize();

        hipMemcpy(bStorage.dat, _bStorage_dev.dat, bStorage.chunks * bStorage.step * sizeof(bElem), hipMemcpyDeviceToHost);

        if (!compareBrick<3>({STRIDE, STRIDE, STRIDE}, phi_new_arr, grid_ptr, phi_new_bri))
            std::cout << "brick don't match" << std::endl;
    }
    return 0;
}

