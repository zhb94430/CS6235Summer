#include "hip/hip_runtime.h"
#include "brick-cuda.h"
#include "head.h"
#include "headcu.h"

#define H2INV  0.8f

__global__ void GSRBKernel(double* phi, double* phi_new, double* rhs, double* alpha, double* beta_i,
                           double* beta_j, double* beta_k, double* lambda, int color)
{
    int i, j, k;
    i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= STRIDE-GZ || i < GZ)
    {
       return;
    }

    for (k=GZ; k<STRIDE-GZ; k++)
    {
        for(j=GZ; j<STRIDE-GZ; j++)
        {

            int ijk = i + j*STRIDE + k*(STRIDE*STRIDE);

            // if ((i+j+k+color) % 2 == 0)
            {
                double helmholtz = alpha[ijk]*phi[ijk]
                                 - H2INV*(
                                       beta_i[ijk+1     ]*( phi[ijk+1     ]-phi[ijk       ] )
                                     - beta_i[ijk       ]*( phi[ijk       ]-phi[ijk-1     ] )
                                     + beta_j[ijk+STRIDE]*( phi[ijk+STRIDE]-phi[ijk       ] )
                                     - beta_j[ijk       ]*( phi[ijk       ]-phi[ijk-STRIDE] )
                                     + beta_k[ijk+(STRIDE*STRIDE) ]*( phi[ijk+(STRIDE*STRIDE) ]-phi[ijk       ] )
                                     - beta_k[ijk       ]*( phi[ijk       ]-phi[ijk-(STRIDE*STRIDE) ] )
                                      );

                phi_new[ijk] = phi[ijk] - lambda[ijk]*(helmholtz-rhs[ijk]);
            }
        }
    }
}

__global__ void
brick_kernel(unsigned (*grid)[STRIDE/TILEJ][STRIDE/TILEI], Brick3D phi, Brick3D alpha, Brick3D beta_i, Brick3D beta_j,
        Brick3D beta_k, Brick3D phi_new, Brick3D lambda, Brick3D rhs) {
    unsigned k = GZ / TILEK + blockIdx.z;
    unsigned j = GZ / TILEJ + blockIdx.y;
    unsigned i = GZ / TILEI + blockIdx.x;

    unsigned b = grid[k][j][i];

    brick("kernel.py", BVEC, (TILEK, TILEJ, TILEI), (BFOLD), b);
}

int main() {
    // allocations
    bElem *c = randomArray({3});
    bElem *c_dev;
    copyToDevice({3}, c_dev, c);

    auto phi_arr = randomArray({STRIDE, STRIDE, STRIDE});
    bElem *phi_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, phi_dev, phi_arr);

    auto phi_new_arr = zeroArray({STRIDE, STRIDE, STRIDE});
    bElem *phi_new_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, phi_new_dev, phi_new_arr);

    auto alpha_arr = randomArray({STRIDE, STRIDE, STRIDE});
    bElem *alpha_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, alpha_dev, alpha_arr);

    auto beta_i_arr = randomArray({STRIDE, STRIDE, STRIDE});
    bElem *beta_i_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, beta_i_dev, beta_i_arr);

    auto beta_j_arr = randomArray({STRIDE, STRIDE, STRIDE});
    bElem *beta_j_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, beta_j_dev, beta_j_arr);

    auto beta_k_arr = randomArray({STRIDE, STRIDE, STRIDE});
    bElem *beta_k_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, beta_k_dev, beta_k_arr);

    auto rhs_arr = randomArray({STRIDE, STRIDE, STRIDE});
    bElem *rhs_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, rhs_dev, rhs_arr);

    auto lambda_arr = randomArray({STRIDE, STRIDE, STRIDE});
    bElem *lambda_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, lambda_dev, lambda_arr);

    // CPU
    {

        for(int k=GZ;k<STRIDE-GZ;k++){
            for(int j=GZ;j<STRIDE-GZ;j++){
                for(int i=GZ;i<STRIDE-GZ;i++){
                    int ijk = i + j*STRIDE + k*STRIDE*STRIDE;

                    { // color signifies red or black case
                      double helmholtz = alpha_arr[ijk]*phi_arr[ijk]
                                       - H2INV*(
                                           beta_i_arr[ijk+1     ]*( phi_arr[ijk+1     ]-phi_arr[ijk       ] )
                                         - beta_i_arr[ijk       ]*( phi_arr[ijk       ]-phi_arr[ijk-1     ] )
                                         + beta_j_arr[ijk+STRIDE]*( phi_arr[ijk+STRIDE]-phi_arr[ijk       ] )
                                         - beta_j_arr[ijk       ]*( phi_arr[ijk       ]-phi_arr[ijk-STRIDE] )
                                         + beta_k_arr[ijk+(STRIDE*STRIDE) ]*( phi_arr[ijk+(STRIDE*STRIDE) ]-phi_arr[ijk       ] )
                                         - beta_k_arr[ijk       ]*( phi_arr[ijk       ]-phi_arr[ijk-(STRIDE*STRIDE) ] )
                                            );

                      phi_new_arr[ijk] = phi_arr[ijk] - lambda_arr[ijk]*(helmholtz-rhs_arr[ijk]);
                    }
                }
            }
        }
    }

    // CUDA
    {
        auto compute = [&]() -> void {
            long numOfThreads = STRIDE;
            // long numOfBlocks = ceil(STRIDE/numOfThreads);
            long numOfBlocks = ceil(STRIDE*STRIDE*STRIDE/(numOfThreads*STRIDE*STRIDE)); // Unroll on i

            dim3 dimBlock(numOfThreads);
            dim3 dimGrid(numOfBlocks);
            
            GSRBKernel<<<dimGrid, dimBlock>>> (phi_dev, phi_new_dev, rhs_dev, alpha_dev, beta_i_dev, 
                                         beta_j_dev , beta_k_dev , lambda_dev, 0);
        };
        
        compute();
        auto phi_new_cuda = zeroArray({STRIDE, STRIDE, STRIDE});
        copyFromDevice({STRIDE, STRIDE, STRIDE}, phi_new_cuda, phi_new_dev);

        if (!compareArray({STRIDE, STRIDE, STRIDE}, (bElem *)phi_new_arr, (bElem *)phi_new_cuda))
            std::cout << "arr don't match" << std::endl;
    }

    hipDeviceSynchronize();

    hipFree(phi_dev);
    hipFree(phi_new_dev);
    hipFree(alpha_dev);
    hipFree(beta_i_dev);
    hipFree(beta_j_dev);
    hipFree(beta_k_dev);
    hipFree(rhs_dev);
    hipFree(lambda_dev);
    
    {
        unsigned *grid_ptr;
        unsigned bSize = TILEK * TILEJ * TILEI;

        auto bInfo = init_grid<3>(grid_ptr, {STRIDE/TILEK, STRIDE/TILEJ, STRIDE/TILEI});

        unsigned *grid_dev;
        copyToDevice({STRIDE/TILEK, STRIDE/TILEJ, STRIDE/TILEI}, grid_dev, grid_ptr);

        auto bStorage = BrickStorage::allocate(bInfo.nbricks, bSize * 8);

        Brick3D phi_bri(&bInfo, &bStorage, 0);
        Brick3D alpha_bri(&bInfo, &bStorage, bSize);
        Brick3D beta_i_bri(&bInfo, &bStorage, bSize * 2);
        Brick3D beta_j_bri(&bInfo, &bStorage, bSize * 3);
        Brick3D beta_k_bri(&bInfo, &bStorage, bSize * 4);
        Brick3D phi_new_bri(&bInfo, &bStorage, bSize * 5);
        Brick3D lambda_bri(&bInfo, &bStorage, bSize * 6);
        Brick3D rhs_bri(&bInfo, &bStorage, bSize * 7);

        BrickInfo<3> *bInfo_dev;
        auto _bInfo_dev = movBrickInfo(bInfo, hipMemcpyHostToDevice);
        {
            unsigned size = sizeof(BrickInfo<3>);
            hipMalloc(&bInfo_dev, size);
            hipMemcpy(bInfo_dev, &_bInfo_dev, size, hipMemcpyHostToDevice);
        }

        copyBrick<3>({STRIDE, STRIDE, STRIDE}, phi_arr, grid_ptr, phi_bri);
        copyBrick<3>({STRIDE, STRIDE, STRIDE}, alpha_arr, grid_ptr, alpha_bri);
        copyBrick<3>({STRIDE, STRIDE, STRIDE}, beta_i_arr, grid_ptr, beta_i_bri);
        copyBrick<3>({STRIDE, STRIDE, STRIDE}, beta_j_arr, grid_ptr, beta_j_bri);
        copyBrick<3>({STRIDE, STRIDE, STRIDE}, beta_k_arr, grid_ptr, beta_k_bri);
        copyBrick<3>({STRIDE, STRIDE, STRIDE}, phi_new_arr, grid_ptr, phi_new_bri);
        copyBrick<3>({STRIDE, STRIDE, STRIDE}, lambda_arr, grid_ptr, lambda_bri);
        copyBrick<3>({STRIDE, STRIDE, STRIDE}, rhs_arr, grid_ptr, rhs_bri);

        BrickStorage *bStorage_dev;
        BrickStorage _bStorage_dev = movBrickStorage(bStorage, hipMemcpyHostToDevice);
        {
            unsigned size = sizeof(BrickStorage);
            hipMalloc(&bStorage_dev, size);
            hipMemcpy(bStorage_dev, &_bStorage_dev, size, hipMemcpyHostToDevice);
        }

        auto compute = [&]() -> void {
            Brick3D phi(bInfo_dev, &_bStorage_dev, 0);
            Brick3D alpha(bInfo_dev, &_bStorage_dev, bSize);
            Brick3D beta_i(bInfo_dev, &_bStorage_dev, bSize * 2);
            Brick3D beta_j(bInfo_dev, &_bStorage_dev, bSize * 3);
            Brick3D beta_k(bInfo_dev, &_bStorage_dev, bSize * 4);
            Brick3D phi_new(bInfo_dev, &_bStorage_dev, bSize * 5);
            Brick3D lambda(bInfo_dev, &bStorage, bSize * 6);
            Brick3D rhs(bInfo_dev, &bStorage, bSize * 7);
           
            phi.bStorage = bStorage_dev;
            alpha.bStorage = bStorage_dev;
            beta_i.bStorage = bStorage_dev;
            beta_j.bStorage = bStorage_dev;
            beta_k.bStorage = bStorage_dev;
            phi_new.bStorage = bStorage_dev;
            lambda.bStorage = bStorage_dev;
            rhs.bStorage = bStorage_dev;

            auto grid = (unsigned (*)[STRIDE/TILEJ][STRIDE/TILEI]) grid_dev;
            dim3 block(N/TILEI, N/TILEJ, N/TILEK), thread(32);
            
            brick_kernel<<< block, thread >>>(grid, phi, alpha, beta_i, beta_j, beta_k, phi_new, lambda, rhs);
        };

        compute();

        hipDeviceSynchronize();

        hipMemcpy(bStorage.dat, _bStorage_dev.dat, bStorage.chunks * bStorage.step * sizeof(bElem), hipMemcpyDeviceToHost);

        if (!compareBrick<3>({STRIDE, STRIDE, STRIDE}, phi_new_arr, grid_ptr, phi_new_bri))
            std::cout << "brick don't match" << std::endl;
    }
    return 0;
}

