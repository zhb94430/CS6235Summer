#include "hip/hip_runtime.h"
#include "brick-cuda.h"
#include "head.h"
#include "headcu.h"

#define x(i, j, k) x_arr[k][j][i]
#define alpha(i, j, k) alpha_arr[k][j][i]
#define beta_i(i, j, k) beta_i_arr[k][j][i]
#define beta_j(i, j, k) beta_j_arr[k][j][i]
#define beta_k(i, j, k) beta_k_arr[k][j][i]
#define out(i, j, k) out_arr[k][j][i]
#define c1 c[0]
#define c2 c[1]
#define h2inv c[2]

__global__ void
arr_kernel(bElem *x_ptr, bElem *alpha_ptr, bElem *beta_i_ptr, bElem *beta_j_ptr, bElem *beta_k_ptr, bElem *out_ptr,
        bElem *c) {
    auto x_arr = (bElem (*)[STRIDE][STRIDE]) x_ptr;
    auto alpha_arr = (bElem (*)[STRIDE][STRIDE]) alpha_ptr;
    auto beta_i_arr = (bElem (*)[STRIDE][STRIDE]) beta_i_ptr;
    auto beta_j_arr = (bElem (*)[STRIDE][STRIDE]) beta_j_ptr;
    auto beta_k_arr = (bElem (*)[STRIDE][STRIDE]) beta_k_ptr;

    auto out_arr = (bElem (*)[STRIDE][STRIDE]) out_ptr;

    #include "arrcusched.h"
    {
        #include "kernel.h"
    }
}

#undef x
#undef alpha
#undef beta_i
#undef beta_j
#undef beta_k
#undef out

__global__ void
brick_kernel(unsigned (*grid)[STRIDE/TILEJ][STRIDE/TILEI], Brick3D x, Brick3D alpha, Brick3D beta_i, Brick3D beta_j,
        Brick3D beta_k, Brick3D out, bElem *c) {
    #include "bricusched.h"
    brick("kernel.py", BVEC, (TILEK, TILEJ, TILEI), (BFOLD), b);
}

int main() {
    // allocations
    bElem *c = randomArray({3});
    bElem *c_dev;
    copyToDevice({3}, c_dev, c);

    auto x_arr = randomArray({STRIDE, STRIDE, STRIDE});
    bElem *x_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, x_dev, x_arr);

    auto alpha_arr = randomArray({STRIDE, STRIDE, STRIDE});
    bElem *alpha_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, alpha_dev, alpha_arr);

    auto beta_i_arr = randomArray({STRIDE, STRIDE, STRIDE});
    bElem *beta_i_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, beta_i_dev, beta_i_arr);

    auto beta_j_arr = randomArray({STRIDE, STRIDE, STRIDE});
    bElem *beta_j_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, beta_j_dev, beta_j_arr);

    auto beta_k_arr = randomArray({STRIDE, STRIDE, STRIDE});
    bElem *beta_k_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, beta_k_dev, beta_k_arr);

    auto out_arr = zeroArray({STRIDE, STRIDE, STRIDE});
    bElem *out_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, out_dev, out_arr);

    {
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(arr_kernel), hipFuncCachePreferL1);

        auto compute = [&]() -> void {
            dim3 block(N/TILEI, N/TILEJ, N/TILEK), thread(_TILEI, _TILEJ, _TILEK);
            
            arr_kernel<<< block, thread >>>(x_dev, alpha_dev, beta_i_dev, beta_j_dev, beta_k_dev, out_dev, c_dev);
        };
        
#ifndef TYPE
        #include "cutiming.h"
#else
        compute();
        copyFromDevice({STRIDE, STRIDE, STRIDE}, out_arr, out_dev);
#endif
    }

    hipDeviceSynchronize();

    hipFree(x_dev);
    hipFree(alpha_dev);
    hipFree(beta_i_dev);
    hipFree(beta_j_dev);
    hipFree(beta_k_dev);
    hipFree(out_dev);

#if TYPE == 1
    {
        unsigned *grid_ptr;
        unsigned bSize = TILEK * TILEJ * TILEI;

        auto bInfo = init_grid<3>(grid_ptr, {STRIDE/TILEK, STRIDE/TILEJ, STRIDE/TILEI});

        unsigned *grid_dev;
        copyToDevice({STRIDE/TILEK, STRIDE/TILEJ, STRIDE/TILEI}, grid_dev, grid_ptr);

        auto bStorage = BrickStorage::allocate(bInfo.nbricks, bSize * 6);

        Brick3D x_bri(&bInfo, &bStorage, 0);
        Brick3D alpha_bri(&bInfo, &bStorage, bSize);
        Brick3D beta_i_bri(&bInfo, &bStorage, bSize * 2);
        Brick3D beta_j_bri(&bInfo, &bStorage, bSize * 3);
        Brick3D beta_k_bri(&bInfo, &bStorage, bSize * 4);
        Brick3D out_bri(&bInfo, &bStorage, bSize * 5);

        BrickInfo<3> *bInfo_dev;
        auto _bInfo_dev = movBrickInfo(bInfo, hipMemcpyHostToDevice);
        {
            unsigned size = sizeof(BrickInfo<3>);
            hipMalloc(&bInfo_dev, size);
            hipMemcpy(bInfo_dev, &_bInfo_dev, size, hipMemcpyHostToDevice);
        }

        copyBrick<3>({STRIDE, STRIDE, STRIDE}, x_arr, grid_ptr, x_bri);
        copyBrick<3>({STRIDE, STRIDE, STRIDE}, alpha_arr, grid_ptr, alpha_bri);
        copyBrick<3>({STRIDE, STRIDE, STRIDE}, beta_i_arr, grid_ptr, beta_i_bri);
        copyBrick<3>({STRIDE, STRIDE, STRIDE}, beta_j_arr, grid_ptr, beta_j_bri);
        copyBrick<3>({STRIDE, STRIDE, STRIDE}, beta_k_arr, grid_ptr, beta_k_bri);

        BrickStorage *bStorage_dev;
        BrickStorage _bStorage_dev = movBrickStorage(bStorage, hipMemcpyHostToDevice);
        {
            unsigned size = sizeof(BrickStorage);
            hipMalloc(&bStorage_dev, size);
            hipMemcpy(bStorage_dev, &_bStorage_dev, size, hipMemcpyHostToDevice);
        }

        hipFuncSetCacheConfig(reinterpret_cast<const void*>(brick_kernel), hipFuncCachePreferL1);

        auto compute = [&]() -> void {
            Brick3D x(bInfo_dev, &_bStorage_dev, 0);
            Brick3D alpha(bInfo_dev, &_bStorage_dev, bSize);
            Brick3D beta_i(bInfo_dev, &_bStorage_dev, bSize * 2);
            Brick3D beta_j(bInfo_dev, &_bStorage_dev, bSize * 3);
            Brick3D beta_k(bInfo_dev, &_bStorage_dev, bSize * 4);
            Brick3D out(bInfo_dev, &_bStorage_dev, bSize * 5);
           
            x.bStorage = bStorage_dev;
            alpha.bStorage = bStorage_dev;
            beta_i.bStorage = bStorage_dev;
            beta_j.bStorage = bStorage_dev;
            beta_k.bStorage = bStorage_dev;
            out.bStorage = bStorage_dev;

            auto grid = (unsigned (*)[STRIDE/TILEJ][STRIDE/TILEI]) grid_dev;
            dim3 block(N/TILEI, N/TILEJ, N/TILEK), thread(32);
            
            brick_kernel<<< block, thread >>>(grid, x, alpha, beta_i, beta_j, beta_k, out, c_dev);
        };

        #include "cutiming.h"

        hipDeviceSynchronize();

        hipMemcpy(bStorage.dat, _bStorage_dev.dat, bStorage.chunks * bStorage.step * sizeof(bElem), hipMemcpyDeviceToHost);

        if (!compareBrick<3>({STRIDE, STRIDE, STRIDE}, out_arr, grid_ptr, out_bri))
            return 1;
    }
#endif
    return 0;
}
