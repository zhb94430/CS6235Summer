#include "hip/hip_runtime.h"
#include "brick-cuda.h"
#include "head.h"
#include "headcu.h"

#define ALP  0.8
#define BET (-0.2)
#define GAM  0.038095238095238 //  4.0/105.0
#define DEL (-0.003571428571429) // -1.0/280.0

#define qu(i, j, k) qu_arr[k][j][i]
#define qv(i, j, k) qv_arr[k][j][i]
#define qw(i, j, k) qw_arr[k][j][i]
#define qpres(i, j, k) qpres_arr[k][j][i]

#define cons_imx(i, j, k) cons_imx_arr[k][j][i]
#define cons_imy(i, j, k) cons_imy_arr[k][j][i]
#define cons_imz(i, j, k) cons_imz_arr[k][j][i]
#define cons_iene(i, j, k) cons_iene_arr[k][j][i]

#define flux_irho(i, j, k) flux_irho_arr[k][j][i]
#define flux_imx(i, j, k) flux_imx_arr[k][j][i]
#define flux_imy(i, j, k) flux_imy_arr[k][j][i]
#define flux_imz(i, j, k) flux_imz_arr[k][j][i]
#define flux_iene(i, j, k) flux_iene_arr[k][j][i]

#define dxinv0 c[0]
#define dxinv1 c[1]
#define dxinv2 c[2]

__global__ void
arr_kernel(
        bElem *qu_ptr, bElem *qv_ptr, bElem *qw_ptr, bElem *qpres_ptr, 
        bElem *cons_imx_ptr, bElem *cons_imy_ptr, bElem *cons_imz_ptr, bElem *cons_iene_ptr, 
        bElem *flux_irho_ptr, bElem *flux_imx_ptr, bElem *flux_imy_ptr, bElem *flux_imz_ptr, bElem *flux_iene_ptr,
        bElem *c) {

    auto qu_arr = (bElem (*)[STRIDE][STRIDE]) qu_ptr;
    auto qv_arr = (bElem (*)[STRIDE][STRIDE]) qv_ptr;
    auto qw_arr = (bElem (*)[STRIDE][STRIDE]) qw_ptr;
    auto qpres_arr = (bElem (*)[STRIDE][STRIDE]) qpres_ptr;

    auto cons_imx_arr = (bElem (*)[STRIDE][STRIDE]) cons_imx_ptr;
    auto cons_imy_arr = (bElem (*)[STRIDE][STRIDE]) cons_imy_ptr;
    auto cons_imz_arr = (bElem (*)[STRIDE][STRIDE]) cons_imz_ptr;
    auto cons_iene_arr = (bElem (*)[STRIDE][STRIDE]) cons_iene_ptr;

    auto flux_irho_arr = (bElem (*)[STRIDE][STRIDE]) flux_irho_ptr;
    auto flux_imx_arr = (bElem (*)[STRIDE][STRIDE]) flux_imx_ptr;
    auto flux_imy_arr = (bElem (*)[STRIDE][STRIDE]) flux_imy_ptr;
    auto flux_imz_arr = (bElem (*)[STRIDE][STRIDE]) flux_imz_ptr;
    auto flux_iene_arr = (bElem (*)[STRIDE][STRIDE]) flux_iene_ptr;

    #include "arrcusched.h"
    {
        #include "kernel.h"
    }
}

#undef qu
#undef qv
#undef qw
#undef qpres

#undef cons_imx
#undef cons_imy
#undef cons_imz
#undef cons_iene

#undef flux_irho
#undef flux_imx
#undef flux_imy
#undef flux_imz
#undef flux_iene

__global__ void
brick_kernel(
        unsigned (*grid)[STRIDE/TILEJ][STRIDE/TILEI],
        Brick3D qu, Brick3D qv, Brick3D qw, Brick3D qpres, 
        Brick3D cons_imx, Brick3D cons_imy, Brick3D cons_imz, Brick3D cons_iene, 
        Brick3D flux_irho, Brick3D flux_imx, Brick3D flux_imy, Brick3D flux_imz, Brick3D flux_iene,
        bElem *c) {
    #include "bricusched.h"
    brick("kernel.py", BVEC, (TILEK, TILEJ, TILEI), (BFOLD), b);
}

int main() {
    // allocations
    bElem *c = randomArray({3});
    bElem *c_dev;
    copyToDevice({3}, c_dev, c);

    auto qu_arr = randomArray({STRIDE, STRIDE, STRIDE});
    bElem *qu_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, qu_dev, qu_arr);

    auto qv_arr = randomArray({STRIDE, STRIDE, STRIDE});
    bElem *qv_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, qv_dev, qv_arr);

    auto qw_arr = randomArray({STRIDE, STRIDE, STRIDE});
    bElem *qw_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, qw_dev, qw_arr);

    auto qpres_arr = randomArray({STRIDE, STRIDE, STRIDE});
    bElem *qpres_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, qpres_dev, qpres_arr);

    auto cons_imx_arr = randomArray({STRIDE, STRIDE, STRIDE});
    bElem *cons_imx_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, cons_imx_dev, cons_imx_arr);

    auto cons_imy_arr = randomArray({STRIDE, STRIDE, STRIDE});
    bElem *cons_imy_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, cons_imy_dev, cons_imy_arr);

    auto cons_imz_arr = randomArray({STRIDE, STRIDE, STRIDE});
    bElem *cons_imz_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, cons_imz_dev, cons_imz_arr);

    auto cons_iene_arr = randomArray({STRIDE, STRIDE, STRIDE});
    bElem *cons_iene_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, cons_iene_dev, cons_iene_arr);

    auto flux_irho_arr = zeroArray({STRIDE, STRIDE, STRIDE});
    bElem *flux_irho_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, flux_irho_dev, flux_irho_arr);

    auto flux_imx_arr = zeroArray({STRIDE, STRIDE, STRIDE});
    bElem *flux_imx_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, flux_imx_dev, flux_imx_arr);

    auto flux_imy_arr = zeroArray({STRIDE, STRIDE, STRIDE});
    bElem *flux_imy_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, flux_imy_dev, flux_imy_arr);

    auto flux_imz_arr = zeroArray({STRIDE, STRIDE, STRIDE});
    bElem *flux_imz_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, flux_imz_dev, flux_imz_arr);

    auto flux_iene_arr = zeroArray({STRIDE, STRIDE, STRIDE});
    bElem *flux_iene_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, flux_iene_dev, flux_iene_arr);

    {
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(arr_kernel), hipFuncCachePreferL1);

        auto compute = [&]() -> void {
            dim3 block(N/TILEI, N/TILEJ, N/TILEK), thread(_TILEI, _TILEJ, _TILEK);
            
            arr_kernel<<< block, thread >>>(qu_dev, qv_dev, qw_dev, qpres_dev, 
                    cons_imx_dev, cons_imy_dev, cons_imz_dev, cons_iene_dev, 
                    flux_irho_dev, flux_imx_dev, flux_imy_dev, flux_imz_dev, flux_iene_dev,
                    c_dev);
        };
        
#ifndef TYPE
        #include "cutiming.h"
#else
        compute();
        copyFromDevice({STRIDE, STRIDE, STRIDE}, flux_iene_arr, flux_iene_dev);
#endif
    }

    hipDeviceSynchronize();

    hipFree(qu_dev);
    hipFree(qv_dev);
    hipFree(qw_dev);
    hipFree(qpres_dev);

    hipFree(cons_imx_dev);
    hipFree(cons_imy_dev);
    hipFree(cons_imz_dev);
    hipFree(cons_iene_dev);

    hipFree(flux_irho_dev);
    hipFree(flux_imx_dev);
    hipFree(flux_imy_dev);
    hipFree(flux_imz_dev);
    hipFree(flux_iene_dev);

#if TYPE == 1
    {
        unsigned *grid_ptr;
        unsigned bSize = TILEK * TILEJ * TILEI;

        auto bInfo = init_grid<3>(grid_ptr, {STRIDE/TILEK, STRIDE/TILEJ, STRIDE/TILEI});

        auto bStorage = BrickStorage::allocate(bInfo.nbricks, bSize * 13);
        Brick3D qu_bri(&bInfo, &bStorage, 0);
        Brick3D qv_bri(&bInfo, &bStorage, bSize);
        Brick3D qw_bri(&bInfo, &bStorage, bSize * 2);
        Brick3D qpres_bri(&bInfo, &bStorage, bSize * 3);

        Brick3D cons_imx_bri(&bInfo, &bStorage, bSize * 4);
        Brick3D cons_imy_bri(&bInfo, &bStorage, bSize * 5);
        Brick3D cons_imz_bri(&bInfo, &bStorage, bSize * 6);
        Brick3D cons_iene_bri(&bInfo, &bStorage, bSize * 7);

        Brick3D flux_irho_bri(&bInfo, &bStorage, bSize * 8);
        Brick3D flux_imx_bri(&bInfo, &bStorage, bSize * 9);
        Brick3D flux_imy_bri(&bInfo, &bStorage, bSize * 10);
        Brick3D flux_imz_bri(&bInfo, &bStorage, bSize * 11);
        Brick3D flux_iene_bri(&bInfo, &bStorage, bSize * 12);

        copyBrick<3>({STRIDE, STRIDE, STRIDE}, (bElem *)qu_arr, grid_ptr, qu_bri);
        copyBrick<3>({STRIDE, STRIDE, STRIDE}, (bElem *)qv_arr, grid_ptr, qv_bri);
        copyBrick<3>({STRIDE, STRIDE, STRIDE}, (bElem *)qw_arr, grid_ptr, qw_bri);
        copyBrick<3>({STRIDE, STRIDE, STRIDE}, (bElem *)qpres_arr, grid_ptr, qpres_bri);

        copyBrick<3>({STRIDE, STRIDE, STRIDE}, (bElem *)cons_imx_arr, grid_ptr, cons_imx_bri);
        copyBrick<3>({STRIDE, STRIDE, STRIDE}, (bElem *)cons_imy_arr, grid_ptr, cons_imy_bri);
        copyBrick<3>({STRIDE, STRIDE, STRIDE}, (bElem *)cons_imz_arr, grid_ptr, cons_imz_bri);
        copyBrick<3>({STRIDE, STRIDE, STRIDE}, (bElem *)cons_iene_arr, grid_ptr, cons_iene_bri);

        unsigned *grid_dev;
        copyToDevice({STRIDE/TILEK, STRIDE/TILEJ, STRIDE/TILEI}, grid_dev, grid_ptr);

        BrickInfo<3> *bInfo_dev;
        auto _bInfo_dev = movBrickInfo(bInfo, hipMemcpyHostToDevice);
        {
            unsigned size = sizeof(BrickInfo<3>);
            hipMalloc(&bInfo_dev, size);
            hipMemcpy(bInfo_dev, &_bInfo_dev, size, hipMemcpyHostToDevice);
        }

        BrickStorage *bStorage_dev;
        BrickStorage _bStorage_dev = movBrickStorage(bStorage, hipMemcpyHostToDevice);
        {
            unsigned size = sizeof(BrickStorage);
            hipMalloc(&bStorage_dev, size);
            hipMemcpy(bStorage_dev, &_bStorage_dev, size, hipMemcpyHostToDevice);
        }

        hipFuncSetCacheConfig(reinterpret_cast<const void*>(brick_kernel), hipFuncCachePreferL1);

        auto compute = [&]() -> void {
            Brick3D qu(bInfo_dev, &_bStorage_dev, 0);
            Brick3D qv(bInfo_dev, &_bStorage_dev, bSize);
            Brick3D qw(bInfo_dev, &_bStorage_dev, bSize * 2);
            Brick3D qpres(bInfo_dev, &_bStorage_dev, bSize * 3);

            Brick3D cons_imx(bInfo_dev, &_bStorage_dev, bSize * 4);
            Brick3D cons_imy(bInfo_dev, &_bStorage_dev, bSize * 5);
            Brick3D cons_imz(bInfo_dev, &_bStorage_dev, bSize * 6);
            Brick3D cons_iene(bInfo_dev, &_bStorage_dev, bSize * 7);

            Brick3D flux_irho(bInfo_dev, &_bStorage_dev, bSize * 8);
            Brick3D flux_imx(bInfo_dev, &_bStorage_dev, bSize * 9);
            Brick3D flux_imy(bInfo_dev, &_bStorage_dev, bSize * 10);
            Brick3D flux_imz(bInfo_dev, &_bStorage_dev, bSize * 11);
            Brick3D flux_iene(bInfo_dev, &_bStorage_dev, bSize * 12);

            qu.bStorage = bStorage_dev;
            qv.bStorage = bStorage_dev;
            qw.bStorage = bStorage_dev;
            qpres.bStorage = bStorage_dev;

            cons_imx.bStorage = bStorage_dev;
            cons_imy.bStorage = bStorage_dev;
            cons_imz.bStorage = bStorage_dev;
            cons_iene.bStorage = bStorage_dev;

            flux_irho.bStorage = bStorage_dev;
            flux_imx.bStorage = bStorage_dev;
            flux_imy.bStorage = bStorage_dev;
            flux_imz.bStorage = bStorage_dev;
            flux_iene.bStorage = bStorage_dev;

            auto grid = (unsigned (*)[STRIDE/TILEJ][STRIDE/TILEI]) grid_dev;
            dim3 block(N/TILEI, N/TILEJ, N/TILEK), thread(32);
            
            brick_kernel<<< block, thread >>>(grid, qu, qv, qw, qpres, 
                    cons_imx, cons_imy, cons_imz, cons_iene, 
                    flux_irho, flux_imx, flux_imy, flux_imz, flux_iene, c_dev);
        };

        #include "cutiming.h"

        hipDeviceSynchronize();

        hipMemcpy(bStorage.dat, _bStorage_dev.dat, bStorage.chunks * bStorage.step * sizeof(bElem), hipMemcpyDeviceToHost);

        if (!compareBrick<3>({STRIDE, STRIDE, STRIDE}, flux_iene_arr, grid_ptr, flux_iene_bri))
            return 1;
    }
#endif
    return 0;
}
