#include "hip/hip_runtime.h"
#include "brick-cuda.h"
#include "head.h"
#include "headcu.h"

#define Ac(i, j, k) Ac_arr[k][j][i]
#define Ap(i, j, k) Ap_arr[k][j][i]
#define Dinv(i, j, k) Dinv_arr[k][j][i]
#define RHS(i, j, k) RHS_arr[k][j][i]
#define out(i, j, k) out_arr[k][j][i]
#define out(i, j, k) out_arr[k][j][i]
#define c1 c[0]
#define c2 c[1]
#define h2inv c[2]

__global__ void
arr_kernel(bElem *Ac_ptr, bElem *Ap_ptr, bElem *Dinv_ptr, bElem *RHS_ptr, bElem *out_ptr,
        bElem *c) {
    auto Ac_arr = (bElem (*)[STRIDE][STRIDE]) Ac_ptr;
    auto Ap_arr = (bElem (*)[STRIDE][STRIDE]) Ap_ptr;
    auto Dinv_arr = (bElem (*)[STRIDE][STRIDE]) Dinv_ptr;
    auto RHS_arr = (bElem (*)[STRIDE][STRIDE]) RHS_ptr;

    auto out_arr = (bElem (*)[STRIDE][STRIDE]) out_ptr;

    #include "arrcusched.h"
    {
        #include "kernel.h"
    }
}

#undef Ac
#undef Ap
#undef Dinv
#undef RHS
#undef out

__global__ void
brick_kernel(unsigned (*grid)[STRIDE/TILEJ][STRIDE/TILEI], Brick3D Ac, Brick3D Ap, Brick3D Dinv, Brick3D RHS, Brick3D out, bElem *c) {
    #include "bricusched.h"
    brick("kernel.py", BVEC, (TILEK, TILEJ, TILEI), (BFOLD), b);
}

int main() {
    // allocations
    bElem *c = randomArray({3});
    bElem *c_dev;
    copyToDevice({3}, c_dev, c);

    auto Ac_arr = randomArray({STRIDE, STRIDE, STRIDE});
    bElem *Ac_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, Ac_dev, Ac_arr);

    auto Ap_arr = randomArray({STRIDE, STRIDE, STRIDE});
    bElem *Ap_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, Ap_dev, Ap_arr);

    auto Dinv_arr = randomArray({STRIDE, STRIDE, STRIDE});
    bElem *Dinv_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, Dinv_dev, Dinv_arr);

    auto RHS_arr = randomArray({STRIDE, STRIDE, STRIDE});
    bElem *RHS_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, RHS_dev, RHS_arr);

    auto out_arr = zeroArray({STRIDE, STRIDE, STRIDE});
    bElem *out_dev;
    copyToDevice({STRIDE, STRIDE, STRIDE}, out_dev, out_arr);

    {
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(arr_kernel), hipFuncCachePreferL1);

        auto compute = [&]() -> void {
            dim3 block(N/TILEI, N/TILEJ, N/TILEK), thread(_TILEI, _TILEJ, _TILEK);
            
            arr_kernel<<< block, thread >>>(Ac_dev, Ap_dev, Dinv_dev, RHS_dev, out_dev, c_dev);
        };
        
#ifndef TYPE
        #include "cutiming.h"
#else
        compute();
        copyFromDevice({STRIDE, STRIDE, STRIDE}, out_arr, out_dev);
#endif
    }

    hipDeviceSynchronize();

    hipFree(Ac_dev);
    hipFree(Ap_dev);
    hipFree(Dinv_dev);
    hipFree(RHS_dev);
    hipFree(out_dev);

#if TYPE == 1
    {
        unsigned *grid_ptr;
        unsigned bSize = TILEK * TILEJ * TILEI;

        auto bInfo = init_grid<3>(grid_ptr, {STRIDE/TILEK, STRIDE/TILEJ, STRIDE/TILEI});

        unsigned *grid_dev;
        copyToDevice({STRIDE/TILEK, STRIDE/TILEJ, STRIDE/TILEI}, grid_dev, grid_ptr);

        auto bStorage = BrickStorage::allocate(bInfo.nbricks, bSize * 5);
        Brick3D Ac_bri(&bInfo, &bStorage, 0);
        Brick3D Ap_bri(&bInfo, &bStorage, bSize);
        Brick3D Dinv_bri(&bInfo, &bStorage, bSize * 2);
        Brick3D RHS_bri(&bInfo, &bStorage, bSize * 3);
        Brick3D out_bri(&bInfo, &bStorage, bSize * 4);

        BrickInfo<3> *bInfo_dev;
        auto _bInfo_dev = movBrickInfo(bInfo, hipMemcpyHostToDevice);
        {
            unsigned size = sizeof(BrickInfo<3>);
            hipMalloc(&bInfo_dev, size);
            hipMemcpy(bInfo_dev, &_bInfo_dev, size, hipMemcpyHostToDevice);
        }

        copyBrick<3>({STRIDE, STRIDE, STRIDE}, Ac_arr, grid_ptr, Ac_bri);
        copyBrick<3>({STRIDE, STRIDE, STRIDE}, Ap_arr, grid_ptr, Ap_bri);
        copyBrick<3>({STRIDE, STRIDE, STRIDE}, Dinv_arr, grid_ptr, Dinv_bri);
        copyBrick<3>({STRIDE, STRIDE, STRIDE}, RHS_arr, grid_ptr, RHS_bri);

        BrickStorage *bStorage_dev;
        BrickStorage _bStorage_dev = movBrickStorage(bStorage, hipMemcpyHostToDevice);
        {
            unsigned size = sizeof(BrickStorage);
            hipMalloc(&bStorage_dev, size);
            hipMemcpy(bStorage_dev, &_bStorage_dev, size, hipMemcpyHostToDevice);
        }

        hipFuncSetCacheConfig(reinterpret_cast<const void*>(brick_kernel), hipFuncCachePreferL1);

        auto compute = [&]() -> void {
            Brick3D Ac(bInfo_dev, &_bStorage_dev, 0);
            Brick3D Ap(bInfo_dev, &_bStorage_dev, bSize);
            Brick3D Dinv(bInfo_dev, &_bStorage_dev, bSize * 2);
            Brick3D RHS(bInfo_dev, &_bStorage_dev, bSize * 3);
            Brick3D out(bInfo_dev, &_bStorage_dev, bSize * 4);

            Ac.bStorage = bStorage_dev;
            Ap.bStorage = bStorage_dev;
            Dinv.bStorage = bStorage_dev;
            RHS.bStorage = bStorage_dev;
            out.bStorage = bStorage_dev;

            auto grid = (unsigned (*)[STRIDE/TILEJ][STRIDE/TILEI]) grid_dev;
            dim3 block(N/TILEI, N/TILEJ, N/TILEK), thread(32);
            
            brick_kernel<<< block, thread >>>(grid, Ac, Ap, Dinv, RHS, out, c_dev);
        };

        #include "cutiming.h"

        hipDeviceSynchronize();

        hipMemcpy(bStorage.dat, _bStorage_dev.dat, bStorage.chunks * bStorage.step * sizeof(bElem), hipMemcpyDeviceToHost);

        if (!compareBrick<3>({STRIDE, STRIDE, STRIDE}, out_arr, grid_ptr, out_bri))
            return 1;
    }
#endif
    return 0;
}
