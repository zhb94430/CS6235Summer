#include "hip/hip_runtime.h"
#include "brick-cuda.h"
#include "head.h"
#include "headcu.h"

#define out(i, j) out_arr[j][i]
#define in(i, j) in_arr[j][i]

__global__ void
arr_kernel(bElem *in_ptr, bElem *out_ptr, bElem *c) {
    auto in_arr = (bElem (*)[STRIDE]) in_ptr;
    auto out_arr = (bElem (*)[STRIDE]) out_ptr;
    #include "arrcusched.h"
    {
        #include "kernel.h"
    }
}

#undef out
#undef in

__global__ void
brick_kernel(unsigned (*grid)[STRIDE/TILEI], Brick2D in, Brick2D out, bElem *c) {
    #include "bricusched.h"
    brick("kernel.py", BVEC, (TILEJ, TILEI), (BFOLD), b);
}

int main() {
    // allocations
    bElem *c = randomArray({17});
    bElem *c_dev;
    copyToDevice({17}, c_dev, c);

    auto in_arr = randomArray({STRIDE, STRIDE});
    bElem *in_dev;
    copyToDevice({STRIDE, STRIDE}, in_dev, in_arr);

    auto out_arr = zeroArray({STRIDE, STRIDE});
    bElem *out_dev;
    copyToDevice({STRIDE, STRIDE}, out_dev, out_arr);

    {
        auto compute = [&]() -> void {
            dim3 block(N/TILEI, N/TILEJ), thread(_TILEI, _TILEJ);
            
            arr_kernel<<< block, thread >>>(in_dev, out_dev, c_dev);
        };
        
#ifndef TYPE
        #include "cutiming.h"
#else
        compute();
#endif
        copyFromDevice({STRIDE, STRIDE}, out_arr, out_dev);
    }

#if TYPE == 1
    {
        unsigned *grid_ptr;
        unsigned bSize = TILEJ * TILEI;

        auto bInfo = init_grid<2>(grid_ptr, {STRIDE/TILEJ, STRIDE/TILEI});

        unsigned *grid_dev;
        copyToDevice({STRIDE/TILEJ, STRIDE/TILEI}, grid_dev, grid_ptr);

        auto bStorage = BrickStorage::allocate(bInfo.nbricks, bSize * 2);
        Brick<Dim<TILEJ, TILEI>, Dim<BFOLD>> in_bri(&bInfo, &bStorage, 0);
        Brick<Dim<TILEJ, TILEI>, Dim<BFOLD>> out_bri(&bInfo, &bStorage, bSize);

        BrickInfo<2> *bInfo_dev;
        auto _bInfo_dev = movBrickInfo(bInfo, hipMemcpyHostToDevice);
        {
            unsigned size = sizeof(BrickInfo<2>);
            hipMalloc(&bInfo_dev, size);
            hipMemcpy(bInfo_dev, &_bInfo_dev, size, hipMemcpyHostToDevice);
        }

        copyBrick<2>({STRIDE, STRIDE}, in_arr, grid_ptr, in_bri);

        BrickStorage *bStorage_dev;
        BrickStorage _bStorage_dev = movBrickStorage(bStorage, hipMemcpyHostToDevice);
        {
            unsigned size = sizeof(BrickStorage);
            hipMalloc(&bStorage_dev, size);
            hipMemcpy(bStorage_dev, &_bStorage_dev, size, hipMemcpyHostToDevice);
        }

        auto compute = [&]() -> void {
            Brick2D bIn(bInfo_dev, &_bStorage_dev, 0);
            Brick2D bOut(bInfo_dev, &_bStorage_dev, bSize);
            bIn.bStorage = bStorage_dev;
            bOut.bStorage = bStorage_dev;
            auto grid = (unsigned (*)[STRIDE/TILEI]) grid_dev;
            dim3 block(N/TILEI, N/TILEJ), thread(32);
            
            brick_kernel<<< block, thread >>>(grid, bIn, bOut, c_dev);
        };

        #include "cutiming.h"

        hipDeviceSynchronize();

        hipMemcpy(bStorage.dat, _bStorage_dev.dat, bStorage.chunks * bStorage.step * sizeof(bElem), hipMemcpyDeviceToHost);

        if (!compareBrick<2>({STRIDE, STRIDE}, out_arr, grid_ptr, out_bri))
            return 1;
    }
#endif
    return 0;
}
